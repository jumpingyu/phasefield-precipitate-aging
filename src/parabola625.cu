#include "hip/hip_runtime.h"
/******************************************************************************
 *                       Code generated with sympy 1.4                        *
 *                                                                            *
 *              See http://www.sympy.org/ for more information.               *
 *                                                                            *
 *                  This file is part of 'PrecipitateAging'                   *
 ******************************************************************************/
#include "parabola625.cuh"
#include <math.h>

__device__ double d_h(double x)
{

	double h_result;
	h_result = x*x*x*(6.0*x*x - 15.0*x + 10.0);
	return h_result;

}

__device__ double d_hprime(double x)
{

	double hprime_result;
	hprime_result = 30.0*x*x*(1.0 - x)*(1.0 - x);
	return hprime_result;

}

__device__ double d_interface_profile(double z)
{

	double interface_profile_result;
	interface_profile_result = 1.0/2.0 - 1.0/2.0*tanh(z);
	return interface_profile_result;

}

__device__ double d_kT()
{

	double kT_result;
	kT_result = 1.5782889043500002e-20;
	return kT_result;

}

__device__ double d_RT()
{

	double RT_result;
	RT_result = 9504.6840941999999;
	return RT_result;

}

__device__ double d_Vm()
{

	double Vm_result;
	Vm_result = 1.0000000000000001e-5;
	return Vm_result;

}

__device__ double d_xe_gam_Cr()
{

	double xe_gam_Cr_result;
	xe_gam_Cr_result = 0.55855269488699388;
	return xe_gam_Cr_result;

}

__device__ double d_xe_gam_Nb()
{

	double xe_gam_Nb_result;
	xe_gam_Nb_result = 0.010717747618334031;
	return xe_gam_Nb_result;

}

__device__ double d_xe_del_Cr()
{

	double xe_del_Cr_result;
	xe_del_Cr_result = 0.031881757028651196;
	return xe_del_Cr_result;

}

__device__ double d_xe_del_Nb()
{

	double xe_del_Nb_result;
	xe_del_Nb_result = 0.16874796371854489;
	return xe_del_Nb_result;

}

__device__ double d_xe_lav_Cr()
{

	double xe_lav_Cr_result;
	xe_lav_Cr_result = 0.44421217837672827;
	return xe_lav_Cr_result;

}

__device__ double d_xe_lav_Nb()
{

	double xe_lav_Nb_result;
	xe_lav_Nb_result = 0.17170586512707406;
	return xe_lav_Nb_result;

}

__device__ double d_matrix_min_Cr()
{

	double matrix_min_Cr_result;
	matrix_min_Cr_result = 0.27939999999999998;
	return matrix_min_Cr_result;

}

__device__ double d_matrix_max_Cr()
{

	double matrix_max_Cr_result;
	matrix_max_Cr_result = 0.32879999999999998;
	return matrix_max_Cr_result;

}

__device__ double d_matrix_min_Nb()
{

	double matrix_min_Nb_result;
	matrix_min_Nb_result = 0.021499999999999998;
	return matrix_min_Nb_result;

}

__device__ double d_matrix_max_Nb()
{

	double matrix_max_Nb_result;
	matrix_max_Nb_result = 0.0269;
	return matrix_max_Nb_result;

}

__device__ double d_enrich_min_Cr()
{

	double enrich_min_Cr_result;
	enrich_min_Cr_result = 0.24729999999999999;
	return enrich_min_Cr_result;

}

__device__ double d_enrich_max_Cr()
{

	double enrich_max_Cr_result;
	enrich_max_Cr_result = 0.29670000000000002;
	return enrich_max_Cr_result;

}

__device__ double d_enrich_min_Nb()
{

	double enrich_min_Nb_result;
	enrich_min_Nb_result = 0.16589999999999999;
	return enrich_min_Nb_result;

}

__device__ double d_enrich_max_Nb()
{

	double enrich_max_Nb_result;
	enrich_max_Nb_result = 0.1726;
	return enrich_max_Nb_result;

}

__device__ double d_xr_gam_Cr(double P_del, double P_lav)
{

	double xr_gam_Cr_result;
	xr_gam_Cr_result = -9.6984079989082258e-10*P_del + 6.3118903213767551e-10*P_lav + 0.55855269488699388;
	return xr_gam_Cr_result;

}

__device__ double d_xr_gam_Nb(double P_del, double P_lav)
{

	double xr_gam_Nb_result;
	xr_gam_Nb_result = 1.0048047396144418e-10*P_del - 1.1106547204101917e-11*P_lav + 0.010717747618334031;
	return xr_gam_Nb_result;

}

__device__ double d_xr_del_Cr(double P_del, double P_lav)
{

	double xr_del_Cr_result;
	xr_del_Cr_result = -6.6194274436412276e-11*P_del - 1.5374687132506466e-10*P_lav + 0.031881757028651196;
	return xr_del_Cr_result;

}

__device__ double d_xr_del_Nb(double P_del, double P_lav)
{

	double xr_del_Nb_result;
	xr_del_Nb_result = -1.9581600075232547e-11*P_del + 3.9815435461767849e-10*P_lav + 0.16874796371854489;
	return xr_del_Nb_result;

}

__device__ double d_xr_lav_Cr(double P_del, double P_lav)
{

	double xr_lav_Cr_result;
	xr_lav_Cr_result = -1.2953044143338369e-9*P_del + 4.3416027339639082e-10*P_lav + 0.44421217837672827;
	return xr_lav_Cr_result;

}

__device__ double d_xr_lav_Nb(double P_del, double P_lav)
{

	double xr_lav_Nb_result;
	xr_lav_Nb_result = 2.6966712244807525e-10*P_del - 6.3173100035954083e-11*P_lav + 0.17170586512707406;
	return xr_lav_Nb_result;

}

__device__ double d_inv_fict_det(double f_del, double f_gam, double f_lav)
{

	double inv_fict_det_result;
	inv_fict_det_result = 10.511929918754895/(0.063619913592430452*f_del*f_del + 0.77472418933378007*f_del*f_gam + 1.0*f_del*f_lav + 0.11406419308318602*f_gam*f_gam + 0.29936120577567832*f_gam*f_lav + 0.076367679948657952*f_lav*f_lav);
	return inv_fict_det_result;

}

__device__ double d_fict_gam_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_gam_Cr_result;
	fict_gam_Cr_result = 0.081719161132759205*INV_DET*(0.94267830414080844*XCR*f_del + 0.13278315265091145*XCR*f_gam + 0.27475106182917869*XCR*f_lav + 0.48142139214471374*XNB*f_del + 1.0*XNB*f_lav - 0.069926420929843164*f_del*f_del - 0.03219039130480722*f_del*f_gam - 0.028703356204519043*f_del*f_lav - 0.028515077359655715*f_gam*f_lav - 0.24409812910594283*f_lav*f_lav);
	return fict_gam_Cr_result;

}

__device__ double d_fict_gam_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_gam_Nb_result;
	fict_gam_Nb_result = -0.010850927847195595*INV_DET*(0.75572285002798056*XCR*f_del - 0.063676254673428662*XCR*f_lav + 0.30738018673156009*XNB*f_del - 1.0*XNB*f_gam - 0.55532654346045196*XNB*f_lav - 0.081941433925801291*f_del*f_del - 0.32945245368491854*f_del*f_gam - 0.38670236006564801*f_del*f_lav + 0.18509555516266776*f_gam*f_lav + 0.11646290035390382*f_lav*f_lav);
	return fict_gam_Nb_result;

}

__device__ double d_fict_del_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_del_Cr_result;
	fict_del_Cr_result = 0.046966148206032843*INV_DET*(0.12886223924609808*XCR*f_del - 0.07101626075977073*XCR*f_gam - 0.20028342359736442*XCR*f_lav - 0.83765336993033834*XNB*f_gam - 1.0*XNB*f_lav + 0.12166908885811056*f_del*f_gam + 0.18246772627205363*f_del*f_lav + 0.056009953433358468*f_gam*f_gam + 0.31729459226818346*f_gam*f_lav + 0.26560576240348166*f_lav*f_lav);
	return fict_del_Cr_result;

}

__device__ double d_fict_del_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_del_Nb_result;
	fict_del_Nb_result = 0.10453655111851477*INV_DET*(0.078444276479274974*XCR*f_gam + 0.20923881653433052*XCR*f_lav + 0.057895185577072183*XNB*f_del + 0.73691813445330934*XNB*f_gam + 1.0*XNB*f_lav - 0.0085055473679878923*f_del*f_gam - 0.031796338023487768*f_del*f_lav - 0.034197271344486177*f_gam*f_gam - 0.24299677574697207*f_gam*f_lav - 0.25292500971253778*f_lav*f_lav);
	return fict_del_Nb_result;

}

__device__ double d_fict_lav_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_lav_Cr_result;
	fict_lav_Cr_result = 0.10453655111851476*INV_DET*(1.0*XCR*f_del + 0.057643074984273575*XCR*f_gam + 0.0694958662006743*XCR*f_lav + 0.44927967972452598*XNB*f_del - 0.78172811574884349*XNB*f_gam - 0.081979041619570731*f_del*f_del - 0.12011579223114696*f_del*f_gam - 0.11933127186562477*f_del*f_lav + 0.022291037694796172*f_gam*f_gam + 0.19081837052380662*f_gam*f_lav);
	return fict_lav_Cr_result;

}

__device__ double d_fict_lav_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_lav_Nb_result;
	fict_lav_Nb_result = -0.02643990820696555*INV_DET*(0.82727610358556924*XCR*f_del + 0.026132709676314116*XCR*f_gam + 0.35577056025509707*XNB*f_del - 0.84918699858043711*XNB*f_gam - 0.27476866079657325*XNB*f_lav - 0.12571448770378987*f_del*f_del - 0.80204383828035919*f_del*f_gam - 1.0*f_del*f_lav - 0.075963142465736502*f_gam*f_gam - 0.047796328138628434*f_gam*f_lav);
	return fict_lav_Nb_result;

}

__device__ double d_s_delta()
{

	double s_delta_result;
	s_delta_result = 0.13;
	return s_delta_result;

}

__device__ double d_s_laves()
{

	double s_laves_result;
	s_laves_result = 0.13;
	return s_laves_result;

}

__device__ double d_g_gam(double XCR, double XNB)
{

	double g_gam_result;
	g_gam_result = 2005672715.5837781*pow(XCR - 0.55855269488699388, 2) + (14703563128.545319*XCR - 8212714809.8900261)*(XNB - 0.010717747618334031) + 62431733279.319511*pow(XNB - 0.010717747618334031, 2);
	return g_gam_result;

}

__device__ double d_g_del(double XCR, double XNB)
{

	double g_del_result;
	g_del_result = 15567985511.489613*pow(XCR - 0.031881757028651196, 2) + (17972099186.595295*XCR - 572982099.56185102)*(XNB - 0.16874796371854489) + 13383100915.719385*pow(XNB - 0.16874796371854489, 2);
	return g_del_result;

}

__device__ double d_g_lav(double XCR, double XNB)
{

	double g_lav_result;
	g_lav_result = 6897850581.6836567*pow(XCR - 0.44421217837672827, 2) + (57317655210.986305*XCR - 25461200480.718456)*(XNB - 0.17170586512707406) + 134480681274.87074*pow(XNB - 0.17170586512707406, 2);
	return g_lav_result;

}

__device__ double d_dg_gam_dxCr(double XCR, double XNB)
{

	double dg_gam_dxCr_result;
	dg_gam_dxCr_result = 4011345431.1675563*XCR + 14703563128.545319*XNB - 2398136879.4032598;
	return dg_gam_dxCr_result;

}

__device__ double d_dg_gam_dxNb(double XCR, double XNB)
{

	double dg_gam_dxNb_result;
	dg_gam_dxNb_result = 14703563128.545319*XCR + 124863466558.63902*XNB - 9550969931.2158108;
	return dg_gam_dxNb_result;

}

__device__ double d_dg_del_dxCr(double XCR, double XNB)
{

	double dg_del_dxCr_result;
	dg_del_dxCr_result = 31135971022.979225*XCR + 17972099186.595295*XNB - 4025424604.4914207;
	return dg_del_dxCr_result;

}

__device__ double d_dg_del_dxNb(double XCR, double XNB)
{

	double dg_del_dxNb_result;
	dg_del_dxNb_result = 17972099186.595295*XCR + 26766201831.43877*XNB - 5089724155.0967312;
	return dg_del_dxNb_result;

}

__device__ double d_dg_lav_dxCr(double XCR, double XNB)
{

	double dg_lav_dxCr_result;
	dg_lav_dxCr_result = 13795701163.367313*XCR + 57317655210.986305*XNB - 15969996041.071507;
	return dg_lav_dxCr_result;

}

__device__ double d_dg_lav_dxNb(double XCR, double XNB)
{

	double dg_lav_dxNb_result;
	dg_lav_dxNb_result = 57317655210.986305*XCR + 268961362549.74149*XNB - 71643443923.07843;
	return dg_lav_dxNb_result;

}

__device__ double d_d2g_gam_dxCrCr()
{

	double d2g_gam_dxCrCr_result;
	d2g_gam_dxCrCr_result = 4011345431.1675563;
	return d2g_gam_dxCrCr_result;

}

__device__ double d_d2g_gam_dxCrNb()
{

	double d2g_gam_dxCrNb_result;
	d2g_gam_dxCrNb_result = 14703563128.545319;
	return d2g_gam_dxCrNb_result;

}

__device__ double d_d2g_gam_dxNbCr()
{

	double d2g_gam_dxNbCr_result;
	d2g_gam_dxNbCr_result = 14703563128.545319;
	return d2g_gam_dxNbCr_result;

}

__device__ double d_d2g_gam_dxNbNb()
{

	double d2g_gam_dxNbNb_result;
	d2g_gam_dxNbNb_result = 124863466558.63902;
	return d2g_gam_dxNbNb_result;

}

__device__ double d_d2g_del_dxCrCr()
{

	double d2g_del_dxCrCr_result;
	d2g_del_dxCrCr_result = 31135971022.979225;
	return d2g_del_dxCrCr_result;

}

__device__ double d_d2g_del_dxCrNb()
{

	double d2g_del_dxCrNb_result;
	d2g_del_dxCrNb_result = 17972099186.595295;
	return d2g_del_dxCrNb_result;

}

__device__ double d_d2g_del_dxNbCr()
{

	double d2g_del_dxNbCr_result;
	d2g_del_dxNbCr_result = 17972099186.595295;
	return d2g_del_dxNbCr_result;

}

__device__ double d_d2g_del_dxNbNb()
{

	double d2g_del_dxNbNb_result;
	d2g_del_dxNbNb_result = 26766201831.43877;
	return d2g_del_dxNbNb_result;

}

__device__ double d_d2g_lav_dxCrCr()
{

	double d2g_lav_dxCrCr_result;
	d2g_lav_dxCrCr_result = 13795701163.367313;
	return d2g_lav_dxCrCr_result;

}

__device__ double d_d2g_lav_dxCrNb()
{

	double d2g_lav_dxCrNb_result;
	d2g_lav_dxCrNb_result = 57317655210.986305;
	return d2g_lav_dxCrNb_result;

}

__device__ double d_d2g_lav_dxNbCr()
{

	double d2g_lav_dxNbCr_result;
	d2g_lav_dxNbCr_result = 57317655210.986305;
	return d2g_lav_dxNbCr_result;

}

__device__ double d_d2g_lav_dxNbNb()
{

	double d2g_lav_dxNbNb_result;
	d2g_lav_dxNbNb_result = 268961362549.74149;
	return d2g_lav_dxNbNb_result;

}

__device__ double d_D_CrCr(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_CrCr_result;
	D_CrCr_result = -2.7980476658284387e-12*pow(XCR, 4)*phi_del - 2.7980476658284387e-12*pow(XCR, 4)*phi_lav - 4.1379136027220276e-13*pow(XCR, 4) - 3.1352143359064083e-12*pow(XCR, 3)*XNB*phi_del - 3.1352143359064083e-12*pow(XCR, 3)*XNB*phi_lav - 1.9305411621056693e-12*pow(XCR, 3)*XNB + 7.2582251443309991e-12*pow(XCR, 3)*phi_del + 7.2582251443309991e-12*pow(XCR, 3)*phi_lav + 1.0733880241977198e-12*pow(XCR, 3) - 3.3716667007796982e-13*pow(XCR, 2)*pow(XNB, 2)*phi_del - 3.3716667007796982e-13*pow(XCR, 2)*pow(XNB, 2)*phi_lav - 1.5167498018334665e-12*pow(XCR, 2)*pow(XNB, 2) + 5.0660713769831958e-12*pow(XCR, 2)*XNB*phi_del + 5.0660713769831958e-12*pow(XCR, 2)*XNB*phi_lav + 3.3852203154225117e-12*pow(XCR, 2)*XNB - 6.6902662177505352e-12*pow(XCR, 2)*phi_del - 6.6902662177505352e-12*pow(XCR, 2)*phi_lav - 9.8939499588778664e-13*pow(XCR, 2) + 2.6872722840769506e-13*XCR*pow(XNB, 2)*phi_del + 2.6872722840769506e-13*XCR*pow(XNB, 2)*phi_lav + 1.2088738496612743e-12*XCR*pow(XNB, 2) - 2.4988159676565734e-12*XCR*XNB*phi_del - 2.4988159676565734e-12*XCR*XNB*phi_lav - 1.5386721816247473e-12*XCR*XNB + 2.2300887392510112e-12*XCR*phi_del + 2.2300887392510112e-12*XCR*phi_lav + 3.2979833196247364e-13*XCR - 1.1448887800131043e-26*XNB*phi_del - 1.1448887800131043e-26*XNB*phi_lav - 5.1503009766791795e-26*XNB + 9.5010974181049266e-26*phi_del + 9.5010974181049266e-26*phi_lav + 1.4050768586718594e-26;
	return D_CrCr_result;

}

__device__ double d_D_CrNb(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_CrNb_result;
	D_CrNb_result = -3.3716667007796982e-13*pow(XCR, 4)*phi_del - 3.3716667007796982e-13*pow(XCR, 4)*phi_lav - 1.5167498018334665e-12*pow(XCR, 4) + 9.7820816736468872e-12*pow(XCR, 3)*XNB*phi_del + 9.7820816736468872e-12*pow(XCR, 3)*XNB*phi_lav - 1.4397072517038633e-11*pow(XCR, 3)*XNB + 8.7462112689409982e-13*pow(XCR, 3)*phi_del + 8.7462112689409982e-13*pow(XCR, 3)*phi_lav + 3.9344975011593433e-12*pow(XCR, 3) + 1.0119248343724856e-11*pow(XCR, 2)*pow(XNB, 2)*phi_del + 1.0119248343724856e-11*pow(XCR, 2)*pow(XNB, 2)*phi_lav - 1.2880322715205165e-11*pow(XCR, 2)*pow(XNB, 2) - 1.7646996655488141e-11*pow(XCR, 2)*XNB*phi_del - 1.7646996655488141e-11*pow(XCR, 2)*XNB*phi_lav + 2.5563893615445791e-11*pow(XCR, 2)*XNB - 8.0618168522379683e-13*pow(XCR, 2)*phi_del - 8.0618168522379683e-13*pow(XCR, 2)*phi_lav - 3.6266215489870243e-12*pow(XCR, 2) - 8.065202768558045e-12*XCR*pow(XNB, 2)*phi_del - 8.065202768558045e-12*XCR*pow(XNB, 2)*phi_lav + 1.0265823200891526e-11*XCR*pow(XNB, 2) + 7.7964755401576761e-12*XCR*XNB*phi_del + 7.7964755401576761e-12*XCR*XNB*phi_lav - 1.1474697050562422e-11*XCR*XNB + 2.6872722840973019e-13*XCR*phi_del + 2.6872722840973019e-13*XCR*phi_lav + 1.2088738496606712e-12*XCR + 3.4361088799843846e-25*XNB*phi_del + 3.4361088799843846e-25*XNB*phi_lav - 4.3736639081722336e-25*XNB + 1.1448887800131042e-26*phi_del + 1.1448887800131042e-26*phi_lav + 5.1503009766791801e-26;
	return D_CrNb_result;

}

__device__ double d_D_NbCr(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_NbCr_result;
	D_NbCr_result = -2.7980476658284383e-12*pow(XCR, 3)*XNB*phi_del - 2.7980476658284383e-12*pow(XCR, 3)*XNB*phi_lav - 4.1379136027220271e-13*pow(XCR, 3)*XNB - 3.1352143359064083e-12*pow(XCR, 2)*pow(XNB, 2)*phi_del - 3.1352143359064083e-12*pow(XCR, 2)*pow(XNB, 2)*phi_lav - 1.9305411621056689e-12*pow(XCR, 2)*pow(XNB, 2) + 5.0281364050823661e-12*pow(XCR, 2)*XNB*phi_del + 5.0281364050823661e-12*pow(XCR, 2)*XNB*phi_lav + 7.4358969223535282e-13*pow(XCR, 2)*XNB - 3.3716667007796982e-13*XCR*pow(XNB, 3)*phi_del - 3.3716667007796982e-13*XCR*pow(XNB, 3)*phi_lav - 1.5167498018334665e-12*XCR*pow(XNB, 3) + 2.5672554093268171e-12*XCR*pow(XNB, 2)*phi_del + 2.5672554093268171e-12*XCR*pow(XNB, 2)*phi_lav + 1.8465481337986408e-12*XCR*pow(XNB, 2) - 2.2300887392502495e-12*XCR*XNB*phi_del - 2.2300887392502495e-12*XCR*XNB*phi_lav - 3.2979833196260607e-13*XCR*XNB - 4.6750799613320131e-25*XCR*phi_del - 4.6750799613320131e-25*XCR*phi_lav - 6.9137767744500761e-26*XCR - 5.099249725448536e-26*pow(XNB, 3)*phi_del - 5.099249725448536e-26*pow(XNB, 3)*phi_lav - 2.2939058622801804e-25*pow(XNB, 3) + 1.6908113649415543e-25*pow(XNB, 2)*phi_del + 1.6908113649415543e-25*pow(XNB, 2)*phi_lav + 7.6061426893702937e-25*pow(XNB, 2) - 2.5533964771926475e-25*XNB*phi_del - 2.5533964771926475e-25*XNB*phi_lav - 7.3529193919553314e-25*XNB + 5.6335035394897672e-26*phi_del + 5.6335035394897672e-26*phi_lav + 2.5342408178048241e-25;
	return D_NbCr_result;

}

__device__ double d_D_NbNb(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_NbNb_result;
	D_NbNb_result = -3.3716667007796982e-13*pow(XCR, 3)*XNB*phi_del - 3.3716667007796982e-13*pow(XCR, 3)*XNB*phi_lav - 1.5167498018334665e-12*pow(XCR, 3)*XNB + 9.7820816736468872e-12*pow(XCR, 2)*pow(XNB, 2)*phi_del + 9.7820816736468872e-12*pow(XCR, 2)*pow(XNB, 2)*phi_lav - 1.4397072517038633e-11*pow(XCR, 2)*pow(XNB, 2) + 6.0589389848635398e-13*pow(XCR, 2)*XNB*phi_del + 6.0589389848635398e-13*pow(XCR, 2)*XNB*phi_lav + 2.7256236514978403e-12*pow(XCR, 2)*XNB + 1.0119248343724856e-11*XCR*pow(XNB, 3)*phi_del + 1.0119248343724856e-11*XCR*pow(XNB, 3)*phi_lav - 1.2880322715205165e-11*XCR*pow(XNB, 3) - 9.8505211153363088e-12*XCR*pow(XNB, 2)*phi_del - 9.8505211153363088e-12*XCR*pow(XNB, 2)*phi_lav + 1.4089196564890813e-11*XCR*pow(XNB, 2) - 2.6872722840794088e-13*XCR*XNB*phi_del - 2.6872722840794088e-13*XCR*XNB*phi_lav - 1.2088738496623801e-12*XCR*XNB - 5.6335035394897672e-26*XCR*phi_del - 5.6335035394897672e-26*XCR*phi_lav - 2.5342408178048241e-25*XCR + 1.5304174142287522e-24*pow(XNB, 3)*phi_del + 1.5304174142287522e-24*pow(XNB, 3)*phi_lav - 1.9479974712212804e-24*pow(XNB, 3) - 5.0745644877292771e-24*pow(XNB, 2)*phi_del - 5.0745644877292771e-24*pow(XNB, 2)*phi_lav + 6.4591781939619209e-24*pow(XNB, 2) + 4.8004314317403787e-24*XNB*phi_del + 4.8004314317403787e-24*XNB*phi_lav - 6.1763226878074476e-24*XNB - 1.6907608734922438e-24*phi_del - 1.6907608734922438e-24*phi_lav + 2.1520912369273906e-24;
	return D_NbNb_result;

}
