#include "hip/hip_runtime.h"
/**
 \file  discretization.cu
 \brief Implementation of discretized equations with CUDA acceleration
 Contains functions for boundary conditions, equations of motion, and nucleation.
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>

#include "data.cuh"
#include "discretization.cuh"
#include "parabola625.cuh"
#include "nucleation.cuh"

/**
 \brief Convenience function for checking CUDA runtime API results
 No-op in release builds.
*/
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

// Convolution mask array on the GPU, allocated in protected memory
__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

// Diffusivity arrays on the GPU, allocated in protected memory
__constant__ fp_t d_DCr[NC];
__constant__ fp_t d_DNb[NC];

// Kinetic parameter arrays on the GPU, allocated in protected memory
__constant__ fp_t d_Kapp[NP];
__constant__ fp_t d_Omeg[NP];
__constant__ fp_t d_Lmob[NP];

float nTiles(int domain_size, int tile_loc, int mask_size)
{
	return ceil(float(domain_size) / float(tile_loc - mask_size + 1));
}

__global__ void boundary_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                fp_t* d_phi_del, fp_t* d_phi_lav,
                                fp_t* d_gam_Cr,  fp_t* d_gam_Nb,
                                const int nx,
                                const int ny,
                                const int nm)
{
	/* determine indices on which to operate */
	const int row = blockDim.y * blockIdx.y + threadIdx.y;
	const int col = blockDim.x * blockIdx.x + threadIdx.x;

	/* apply no-flux boundary conditions: inside to out, sequence matters */

	for (int offset = 0; offset < nm / 2; offset++) {
		const int ilo = nm / 2 - offset;
		const int ihi = nx - 1 - nm / 2 + offset;
		const int jlo = nm / 2 - offset;
		const int jhi = ny - 1 - nm / 2 + offset;

		if (ilo - 1 == col && row < ny) {
			/* left condition */
			d_conc_Cr[row * nx + ilo - 1] = d_conc_Cr[row * nx + ilo];
			d_conc_Nb[row * nx + ilo - 1] = d_conc_Nb[row * nx + ilo];
			d_phi_del[row * nx + ilo - 1] = d_phi_del[row * nx + ilo];
			d_phi_lav[row * nx + ilo - 1] = d_phi_lav[row * nx + ilo];
			d_gam_Cr[ row * nx + ilo - 1] = d_gam_Cr[ row * nx + ilo];
			d_gam_Nb[ row * nx + ilo - 1] = d_gam_Nb[ row * nx + ilo];
		}
		if (ihi + 1 == col && row < ny) {
			/* right condition */
			d_conc_Cr[row * nx + ihi + 1] = d_conc_Cr[row * nx + ihi];
			d_conc_Nb[row * nx + ihi + 1] = d_conc_Nb[row * nx + ihi];
			d_phi_del[row * nx + ihi + 1] = d_phi_del[row * nx + ihi];
			d_phi_lav[row * nx + ihi + 1] = d_phi_lav[row * nx + ihi];
			d_gam_Cr[ row * nx + ihi + 1] = d_gam_Cr[ row * nx + ihi];
			d_gam_Nb[ row * nx + ihi + 1] = d_gam_Nb[ row * nx + ihi];
		}
		if (jlo - 1 == row && col < nx) {
			/* bottom condition */
			d_conc_Cr[(jlo - 1) * nx + col] = d_conc_Cr[jlo * nx + col];
			d_conc_Nb[(jlo - 1) * nx + col] = d_conc_Nb[jlo * nx + col];
			d_phi_del[(jlo - 1) * nx + col] = d_phi_del[jlo * nx + col];
			d_phi_lav[(jlo - 1) * nx + col] = d_phi_lav[jlo * nx + col];
			d_gam_Cr[ (jlo - 1) * nx + col] = d_gam_Cr[ jlo * nx + col];
			d_gam_Nb[ (jlo - 1) * nx + col] = d_gam_Nb[ jlo * nx + col];
		}
		if (jhi + 1 == row && col < nx) {
			/* top condition */
			d_conc_Cr[(jhi + 1) * nx + col] = d_conc_Cr[jhi * nx + col];
			d_conc_Nb[(jhi + 1) * nx + col] = d_conc_Nb[jhi * nx + col];
			d_phi_del[(jhi + 1) * nx + col] = d_phi_del[jhi * nx + col];
			d_phi_lav[(jhi + 1) * nx + col] = d_phi_lav[jhi * nx + col];
			d_gam_Cr[ (jhi + 1) * nx + col] = d_gam_Cr[ jhi * nx + col];
			d_gam_Nb[ (jhi + 1) * nx + col] = d_gam_Nb[ jhi * nx + col];
		}

		__syncthreads();
	}
}

void device_boundaries(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny, nm
	);
}

void device_laplacian_boundaries(struct CudaData* dev,
                                 const int nx, const int ny, const int nm,
                                 const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->lap_gam_Cr,  dev->lap_gam_Nb,
	    nx, ny, nm
	);
}

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new,
                                   const int nx, const int ny, const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm / 2;
	const int src_y = dst_y - nm / 2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y + j) + til_x + i];
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)],
		         NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_conc_new[nx * dst_y + dst_x] = value;
		}
	}
}

void device_laplacian(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->conc_Cr_old, dev->conc_Cr_new, nx, ny, nm);
	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->conc_Nb_old, dev->conc_Nb_new, nx, ny, nm);

	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->gam_Cr, dev->lap_gam_Cr, nx, ny, nm);
	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->gam_Nb, dev->lap_gam_Nb, nx, ny, nm);
}

__device__ void composition_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                                   const fp_t& lap_gam_Cr,  const fp_t& lap_gam_Nb,
                                   fp_t& conc_Cr_new,       fp_t& conc_Nb_new,
                                   const fp_t dt)
{
	/* Cahn-Hilliard equations of motion for composition */

	const fp_t lap_mu_Cr = d_DCr[0] * lap_gam_Cr
	                     + d_DCr[1] * lap_gam_Nb;
	const fp_t lap_mu_Nb = d_DNb[0] * lap_gam_Cr
	                     + d_DNb[1] * lap_gam_Nb;

	conc_Cr_new = conc_Cr_old + dt * lap_mu_Cr;
	conc_Nb_new = conc_Nb_old + dt * lap_mu_Nb;
}

__global__ void cahn_hilliard_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                     fp_t* d_lap_gam_Cr,  fp_t* d_lap_gam_Nb,
                                     fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                     const int nx, const int ny, const int nm,
                                     const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		/* Cahn-Hilliard equations of motion for composition */
		composition_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx],
		                   d_lap_gam_Cr[idx],  d_lap_gam_Nb[idx],
		                   d_conc_Cr_new[idx], d_conc_Nb_new[idx],
		                   dt);
    }
}

__device__ void delta_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                             fp_t& phi_del_new,
                             const fp_t inv_fict_det,
                             const fp_t f_del,       const fp_t f_lav,
                             const fp_t dgGdxCr,     const fp_t dgGdxNb,
                             const fp_t gam_Cr,      const fp_t gam_Nb,
                             const fp_t gam_nrg,     const fp_t alpha,
                             const fp_t dt)
{
	// Derivation: TKR5p281, Eqn. (14)

	const fp_t f_gam = 1. - f_del - f_lav;
	const fp_t del_Cr = d_fict_del_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t del_Nb = d_fict_del_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t del_nrg = d_g_del(del_Cr, del_Nb);

	/* pressure */
	const fp_t P_del = gam_nrg - del_nrg - dgGdxCr * (gam_Cr - del_Cr) - dgGdxNb * (gam_Nb - del_Nb);

	/* variational derivative */
	const fp_t dFdPhi_del = -d_hprime(phi_del_old) * P_del
                            + 2. * d_Omeg[0] * phi_del_old * (phi_del_old - 1.) * (2. * phi_del_old - 1.)
	                        + 2. * alpha * phi_del_old * phi_lav_old * phi_lav_old
	                        - d_Kapp[0] * phi_del_new;

	/* Allen-Cahn equation of motion for delta phase */
	phi_del_new = phi_del_old - dt * d_Lmob[0] * dFdPhi_del;
}

__device__ void laves_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                             fp_t& phi_lav_new,
                             const fp_t inv_fict_det,
                             const fp_t f_del,       const fp_t f_lav,
                             const fp_t dgGdxCr,     const fp_t dgGdxNb,
                             const fp_t gam_Cr,      const fp_t gam_Nb,
                             const fp_t gam_nrg,     const fp_t alpha,
                             const fp_t dt)
{
	// Derivation: TKR5p281, Eqn. (14)

	const fp_t f_gam = 1. - f_del - f_lav;
	const fp_t lav_Cr = d_fict_lav_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t lav_Nb = d_fict_lav_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

	/* pressure */
	const fp_t P_lav = gam_nrg - lav_nrg - dgGdxCr * (gam_Cr - lav_Cr) - dgGdxNb * (gam_Nb - lav_Nb);

	/* variational derivative */
	const fp_t dFdPhi_lav = -d_hprime(phi_lav_old) * P_lav
	                        + 2. * d_Omeg[1] * phi_lav_old * (phi_lav_old - 1.) * (2. * phi_lav_old - 1.)
	                        + 2. * alpha * phi_lav_old * phi_del_old * phi_del_old
	                        - d_Kapp[1] * phi_lav_new;

	/* Allen-Cahn equation of motion for Laves phase */
	phi_lav_new = phi_lav_old - dt * d_Lmob[1] * dFdPhi_lav;
}

__global__ void allen_cahn_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                  fp_t* d_phi_del_old, fp_t* d_phi_lav_old,
                                  fp_t* d_lap_gam_Cr,  fp_t* d_lap_gam_Nb,
                                  fp_t* d_phi_del_new, fp_t* d_phi_lav_new,
                                  fp_t* d_gam_Cr,      fp_t* d_gam_Nb,
                                  const int nx, const int ny, const int nm,
                                  const fp_t alpha,
                                  const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1. - f_del - f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr[idx], d_gam_Nb[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr[idx], d_gam_Nb[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr[idx], d_gam_Nb[idx]);

		/* Allen-Cahn equations of motion for phase */
		delta_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_del_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
		             d_gam_Cr[idx], d_gam_Nb[idx], gam_nrg, alpha, dt);

		laves_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_lav_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
		             d_gam_Cr[idx], d_gam_Nb[idx], gam_nrg, alpha, dt);
	}
}

void device_evolution(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t alpha,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	cahn_hilliard_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->lap_gam_Cr,  dev->lap_gam_Nb,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    nx, ny, nm,
	    dt);

	allen_cahn_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->lap_gam_Cr,  dev->lap_gam_Nb,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny, nm,
	    alpha,
	    dt);
}

__global__ void fictitious_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                  fp_t* d_phi_del, fp_t* d_phi_lav,
                                  fp_t* d_gam_Cr,  fp_t* d_gam_Nb,
                                  const int nx, const int ny)
{
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del[idx]);
		const fp_t f_lav = d_h(d_phi_lav[idx]);
		const fp_t f_gam = 1. - f_del - f_lav;
		const fp_t inv_fict_det = d_inv_fict_det(f_del, f_gam, f_lav);

		d_gam_Cr[idx] = d_fict_gam_Cr(inv_fict_det, d_conc_Cr[idx], d_conc_Nb[idx],
		                              f_del, f_gam, f_lav);
		d_gam_Nb[idx] = d_fict_gam_Nb(inv_fict_det, d_conc_Cr[idx], d_conc_Nb[idx],
		                              f_del, f_gam, f_lav);
	}
}

void device_fictitious(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	fictitious_kernel <<< num_tiles, tile_size>>>(
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny);
}

__global__ void init_prng_kernel(hiprandState* d_prng, const int nx, const int ny)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	if (x < nx && y < ny)
		hiprand_init((unsigned long long)clock() + idx, x, 0, &(d_prng[idx]));
}

void device_init_prng(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	init_prng_kernel <<< num_tiles, tile_size>>> (
	    dev->prng, nx, ny);
}

__device__ void embed_OPC_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                 fp_t* d_phi_del, fp_t* d_phi_lav,
                                 const int nx, const int ny,
                                 const int x, const int y, const int idx,
                                 const fp_t xCr,
                                 const fp_t xNb,
                                 const fp_t par_xe_Cr,
                                 const fp_t par_xe_Nb,
                                 const fp_t R_precip)
{
	const fp_t R_depletion_Cr = fp_t(R_precip) * sqrt((par_xe_Cr - xCr) / (xCr - d_xe_gam_Cr()));
	const fp_t R_depletion_Nb = fp_t(R_precip) * sqrt((par_xe_Nb - xNb) / (xNb - d_xe_gam_Nb()));

	for (int i = -R_precip; i < R_precip; i++) {
		for (int j = -R_precip; j < R_precip; j++) {
			const int idn = nx * (y + j) + (x + i);
			const fp_t r = sqrt(fp_t(i * i + j * j));
			if (idn >= 0 && idn < nx * ny) {
				if (r <= R_precip) {
					d_conc_Cr[idn] = par_xe_Cr;
					d_conc_Nb[idn] = par_xe_Nb;
					d_phi_del[idn] = 1.;
				} else {
					if (r <= R_depletion_Cr)
						d_conc_Cr[idn] = d_xe_gam_Cr();
					if (r <= R_depletion_Nb)
						d_conc_Nb[idn] = d_xe_gam_Nb();
				}
			}
		}
	}
}

__global__ void nucleation_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                  fp_t* d_phi_del, fp_t* d_phi_lav,
                                  hiprandState* d_prng,
                                  const int nx, const int ny, const int nm,
                                  const fp_t D_CrCr, const fp_t D_NbNb,
                                  const fp_t sigma_del, const fp_t sigma_lav,
                                  const fp_t lattice_const,
                                  const fp_t ifce_width,
                                  const fp_t dx, const fp_t dy, const fp_t dz,
                                  const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const fp_t dV = dx * dy * dz;
	const fp_t Vatom = 0.25 * lattice_const * lattice_const * lattice_const; // m³/atom, assuming FCC
	const fp_t n_gam = dV / Vatom; // atoms, assuming FCC

	fp_t phi_pre = 0.;
	fp_t dG_chem = 0.;
	fp_t R_precip, R_star;
	fp_t P_nuc;
	fp_t rand_pre;

	// Scan neighborhood for existing precipitates
	if (x < nx && y < ny) {
		const fp_t R = 1.75e-9 / dx;

		for (int i = -R; i < R; i++) {
			for (int j = -R; j < R; j++) {
				const int idn = nx * (y + j) + (x + i);
				const fp_t r = sqrt(fp_t(i * i + j * j));
				if (idn >= 0 &&
				    idn < nx * ny &&
				    i * i + j * j < R * R)
					phi_pre = max(phi_pre, d_h(d_phi_del[idn])
					              + d_h(d_phi_lav[idn]));
			}
		}
	}
	__syncthreads();

	if (x < nx && y < ny && phi_pre < 1e-10) {
		const int idx = nx * y + x;
		const fp_t xCr = d_conc_Cr[idx];
		const fp_t xNb = d_conc_Nb[idx];

		// Test a delta particle
		d_nucleation_driving_force_delta(xCr, xNb, &dG_chem);
		d_nucleation_probability_sphere(xCr, xNb,
		                                dG_chem,
		                                D_CrCr, D_NbNb,
		                                sigma_del,
		                                Vatom,
		                                n_gam,
		                                dV, dt,
		                                &R_star,
		                                &P_nuc);
		if (R_star > 0.) {
			R_precip = R_star / dx;
			rand_pre = P_nuc - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

			if (rand_pre > 0)
				embed_OPC_kernel(d_conc_Cr, d_conc_Nb,
				                 d_phi_del, d_phi_lav,
				                 nx, ny,
				                 x, y, idx,
				                 xCr, xNb,
				                 d_xe_del_Cr(), d_xe_del_Nb(),
				                 R_precip);
		}

		// Test a Laves particle
		d_nucleation_driving_force_laves(xCr, xNb, &dG_chem);
		d_nucleation_probability_sphere(xCr, xNb,
		                                dG_chem,
		                                D_CrCr, D_NbNb,
		                                sigma_lav,
		                                Vatom,
		                                n_gam,
		                                dV, dt,
		                                &R_star,
		                                &P_nuc);
		if (R_star > 0.) {
			R_precip = R_star / dx;
			rand_pre = P_nuc - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

			if (rand_pre > 0)
				embed_OPC_kernel(d_conc_Cr, d_conc_Nb,
				                 d_phi_lav, d_phi_lav,
				                 nx, ny,
				                 x, y, idx,
				                 xCr, xNb,
				                 d_xe_lav_Cr(), d_xe_lav_Nb(),
				                 R_precip);
		}
	}
}

void device_nucleation(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by,
                       const fp_t* D_Cr, const fp_t* D_Nb,
                       const fp_t sigma_del, const fp_t sigma_lav,
                       const fp_t lattice_const, const fp_t ifce_width,
                       const fp_t dx, const fp_t dy, const fp_t dz,
                       const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	nucleation_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->prng,
	    nx, ny, nm,
	    D_Cr[0], D_Nb[1],
	    sigma_del, sigma_lav,
	    lattice_const, ifce_width,
	    dx, dy, dz, dt);
}

__global__ void dataviz_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb, fp_t* d_conc_Ni,
                               fp_t* d_phi_del, fp_t* d_phi_lav, fp_t* d_phi,
                               const int nx, const int ny)
{
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		d_conc_Ni[idx] = 1. - d_conc_Cr[idx] - d_conc_Nb[idx];
		d_phi[idx] = d_h(d_phi_del[idx]) + d_h(d_phi_lav[idx]);
    }
}

void device_dataviz(struct CudaData* dev, struct HostData* host,
                    const int nx, const int ny, const int nm,
                    const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	dataviz_kernel <<< num_tiles, tile_size>>>(
	    dev->conc_Cr_old, dev->conc_Nb_old, dev->conc_Ni,
	    dev->phi_del_old, dev->phi_lav_old, dev->phi,
	    nx, ny);

	hipMemcpy(host->conc_Ni[0], dev->conc_Ni, nx * ny * sizeof(fp_t),
	                hipMemcpyDeviceToHost);
	hipMemcpy(host->phi[0], dev->phi, nx * ny * sizeof(fp_t),
	                hipMemcpyDeviceToHost);
}
