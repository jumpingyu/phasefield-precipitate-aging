#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "cuda_data.h"
#include "numerics.h"
#include "mesh.h"

#include "cuda_kernels.cuh"
#include "parabola625.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new,
                                   const int nx, const int ny, const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm/2;
	const int src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)], NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_conc_new[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__global__ void evolution_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                 fp_t* d_phi_del_old,
                                 fp_t* d_phi_lav_old,
                                 fp_t* d_gam_Cr_old,  fp_t* d_gam_Nb_old,
                                 fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                 fp_t* d_phi_del_new,
                                 fp_t* d_phi_lav_new,
                                 fp_t* d_gam_Cr_new,  fp_t* d_gam_Nb_new,
                                 const int nx, const int ny, const int nm,
                                 const fp_t D_CrCr, const fp_t D_CrNb,
                                 const fp_t D_NbCr, const fp_t D_NbNb,
                                 const fp_t alpha, const fp_t kappa, const fp_t omega,
                                 const fp_t M_del, const fp_t M_lav,
                                 const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;

	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		const int idx = nx * y + x;

		const fp_t xCr = d_conc_Cr_old[idx];
		const fp_t xNb = d_conc_Nb_old[idx];
		const fp_t phi_del = d_phi_del_old[idx];
		const fp_t phi_lav = d_phi_lav_old[idx];
		const fp_t f_del = d_h(phi_del);
		const fp_t f_lav = d_h(phi_lav);
		const fp_t f_gam = 1. - f_del - f_lav;

		const fp_t lap_Cr = d_gam_Cr_new[idx];
		const fp_t lap_Nb = d_gam_Nb_new[idx];
		const fp_t lap_del = d_phi_del_new[idx];
		const fp_t lap_lav = d_phi_lav_new[idx];

		/* compute fictitious compositions */
		const fp_t gam_Cr = d_gam_Cr_old[idx];
		const fp_t gam_Nb = d_gam_Nb_old[idx];
		const fp_t del_Cr = d_fict_del_Cr(xCr, xNb, f_del, f_gam, f_lav);
		const fp_t del_Nb = d_fict_del_Nb(xCr, xNb, f_del, f_gam, f_lav);
		const fp_t lav_Cr = d_fict_lav_Cr(xCr, xNb, f_del, f_gam, f_lav);
		const fp_t lav_Nb = d_fict_lav_Nb(xCr, xNb, f_del, f_gam, f_lav);

		/* pure phase energies */
		const fp_t gam_nrg = d_g_gam(gam_Cr, gam_Nb);
		const fp_t del_nrg = d_g_del(del_Cr, del_Nb);
		const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(gam_Cr, gam_Nb);
		const fp_t dgGdxNb = d_dg_gam_dxNb(gam_Cr, gam_Nb);

		/* pressure */
        const fp_t sumPhiSq = phi_del * phi_del + phi_lav * phi_lav;

		const fp_t P_del = gam_nrg - del_nrg
		                 - dgGdxCr * (gam_Cr - del_Cr)
		                 - dgGdxNb * (gam_Nb - del_Nb);
		const fp_t P_lav = gam_nrg - lav_nrg
		                 - dgGdxCr * (gam_Cr - lav_Cr)
		                 - dgGdxNb * (gam_Nb - lav_Nb);

		/* variational derivatives */
		const fp_t dFdPhi_del = -d_hprime(phi_del) * P_del
		                      + 2. * omega * phi_del * (phi_del - 1.) * (2. * phi_del - 1.)
		                      + 2. * alpha * phi_del * (sumPhiSq - phi_del * phi_del)
		                      - kappa * lap_del;
		const fp_t dFdPhi_lav = -d_hprime(phi_lav) * P_lav
		                      + 2. * omega * phi_lav * (phi_lav - 1.) * (2. * phi_lav - 1.)
		                      + 2. * alpha * phi_lav * (sumPhiSq - phi_lav * phi_lav)
		                      - kappa * lap_lav;

		/* Cahn-Hilliard equations of motion for composition */
		const fp_t lap_mu_Cr = D_CrCr * lap_Cr
		                     + D_NbCr * lap_Nb;
		const fp_t lap_mu_Nb = D_CrNb * lap_Cr
		                     + D_NbNb * lap_Nb;

		const fp_t conc_Cr_new = xCr + dt * lap_mu_Cr;
		const fp_t conc_Nb_new = xNb + dt * lap_mu_Nb;
		d_conc_Cr_new[idx] = conc_Cr_new;
		d_conc_Nb_new[idx] = conc_Nb_new;

		/* Allen-Cahn equations of motion for phase */
		const fp_t phi_del_new = phi_del - dt * M_del * dFdPhi_del;
		const fp_t phi_lav_new = phi_lav - dt * M_lav * dFdPhi_lav;
		d_phi_del_new[idx] = phi_del_new;
		d_phi_lav_new[idx] = phi_lav_new;

		/* fictitious compositions */
		const fp_t f_del_new = d_h(phi_del_new);
		const fp_t f_lav_new = d_h(phi_lav_new);
		const fp_t f_gam_new = 1. - f_del_new - f_lav_new;
		d_gam_Cr_new[idx] = d_fict_gam_Cr(conc_Cr_new, conc_Nb_new,
		                                  f_del_new, f_gam_new, f_lav_new);
		d_gam_Nb_new[idx] = d_fict_gam_Nb(conc_Cr_new, conc_Nb_new,
		                                  f_del_new, f_gam_new, f_lav_new);
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void device_boundaries(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old,
	    dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    nx, ny, nm
	);
}

void device_laplacian_boundaries(struct CudaData* dev,
                            const int nx, const int ny, const int nm,
                            const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new,
	    dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm
	);
}

void device_laplacian(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Cr_old, dev->conc_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Nb_old, dev->conc_Nb_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Cr_old, dev->gam_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Nb_old, dev->gam_Nb_new, nx, ny, nm);
}

void device_evolution(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t D_CrCr, const fp_t D_CrNb,
                      const fp_t D_NbCr, const fp_t D_NbNb,
                      const fp_t alpha, const fp_t kappa, const fp_t omega,
                      const fp_t M_del, const fp_t M_lav,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	evolution_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm,
	    D_CrCr, D_CrNb,
	    D_NbCr, D_NbNb,
	    alpha, kappa, omega,
	    M_del, M_lav,
	    dt);
}

void read_out_result(struct CudaData* dev, struct HostData* host, const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Cr_new[0], dev->gam_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Nb_new[0], dev->gam_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}
