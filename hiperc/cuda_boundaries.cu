#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_boundaries.cu
 \brief Implementation of boundary condition functions with OpenMP threading
*/

#include <math.h>
#include <omp.h>

extern "C" {
#include "boundaries.h"
}

#include "cuda_kernels.cuh"

__global__ void boundary_kernel(fp_t* d_conc,
                                const int nx,
                                const int ny,
                                const int nm)
{
	/* determine indices on which to operate */
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int row = blockDim.y * blockIdx.y + ty;
	const int col = blockDim.x * blockIdx.x + tx;

	/* apply no-flux boundary conditions: inside to out, sequence matters */

	for (int offset = 0; offset < nm/2; offset++) {
		const int ilo = nm/2 - offset;
		const int ihi = nx - 1 - nm/2 + offset;
		const int jlo = nm/2 - offset;
		const int jhi = ny - 1 - nm/2 + offset;

		if (ilo-1 == col && row < ny) {
			d_conc[row * nx + ilo-1] = d_conc[row * nx + ilo]; /* left condition */
		}
		if (ihi+1 == col && row < ny) {
			d_conc[row * nx + ihi+1] = d_conc[row * nx + ihi]; /* right condition */
		}
		if (jlo-1 == row && col < nx) {
			d_conc[(jlo-1) * nx + col] = d_conc[jlo * nx + col]; /* bottom condition */
		}
		if (jhi+1 == row && col < nx) {
			d_conc[(jhi+1) * nx + col] = d_conc[jhi * nx + col]; /* top condition */
		}

		__syncthreads();
	}
}
