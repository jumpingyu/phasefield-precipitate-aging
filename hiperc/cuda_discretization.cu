#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

extern "C" {
#include "cuda_data.h"
#include "boundaries.h"
#include "numerics.h"
#include "mesh.h"
#include "timer.h"
}

#include "cuda_kernels.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__device__ fp_t dfdc(const fp_t C)
{
  /* do something */
}

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_lap, const fp_t kappa,
                                   const int nx, const int ny, const int nm)
{
	int dst_x, dst_y, dst_nx, dst_ny;
	int src_x, src_y, src_nx, src_ny;
	int til_x, til_y, til_nx;
	fp_t value=0.;

	/* source and tile width include the halo cells */
	src_nx = blockDim.x;
	src_ny = blockDim.y;
	til_nx = src_nx;

	/* destination width excludes the halo cells */
	dst_nx = src_nx - nm + 1;
	dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	til_x = threadIdx.x;
	til_y = threadIdx.y;

	dst_x = blockIdx.x * dst_nx + til_x;
	dst_y = blockIdx.y * dst_ny + til_y;

	src_x = dst_x - nm/2;
	src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
        /* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)], NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
          d_conc_lap[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__global__ void divergence_kernel(fp_t* d_conc_lap, fp_t* d_conc_div,
                                  const int nx, const int ny, const int nm)
{
	int dst_x, dst_y, dst_nx, dst_ny;
	int src_x, src_y, src_nx, src_ny;
	int til_x, til_y, til_nx;
	fp_t value=0.;

	/* source and tile width include the halo cells */
	src_nx = blockDim.x;
	src_ny = blockDim.y;
	til_nx = src_nx;

	/* destination width excludes the halo cells */
	dst_nx = src_nx - nm + 1;
	dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	til_x = threadIdx.x;
	til_y = threadIdx.y;

	dst_x = blockIdx.x * dst_nx + til_x;
	dst_y = blockIdx.y * dst_ny + til_y;

	src_x = dst_x - nm/2;
	src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_lap[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		if (dst_y < ny && dst_x < nx) {
			d_conc_div[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}


__global__ void diffusion_kernel(fp_t* d_conc_old, fp_t* d_conc_div, fp_t* d_conc_new,
                                 const int nx, const int ny, const int nm,
                                 const fp_t D, const fp_t dt)
{
	int thr_x, thr_y, x, y;

	/* determine indices on which to operate */
	thr_x = threadIdx.x;
	thr_y = threadIdx.y;

	x = blockDim.x * blockIdx.x + thr_x;
	y = blockDim.y * blockIdx.y + thr_y;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
      /* do something */
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void device_boundaries(fp_t* conc,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    conc, nx, ny, nm
	);
}

void device_laplacian(fp_t* conc_old, fp_t* conc_lap, const fp_t kappa,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
    	conc_old, conc_lap, kappa, nx, ny, nm
	);
}

void device_divergence(fp_t* conc_lap, fp_t* conc_div,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	divergence_kernel<<<num_tiles,tile_size,buf_size>>> (
		conc_lap, conc_div, nx, ny, nm
	);
}

void device_composition(fp_t* conc_old, fp_t* conc_lap, fp_t* conc_new,
                        const int nx, const int ny, const int nm,
                        const int bx, const int by,
                        const fp_t D, const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	diffusion_kernel<<<num_tiles,tile_size>>> (
	    conc_old, conc_lap, conc_new, nx, ny, nm, D, dt
	);
}

void read_out_result(fp_t** conc, fp_t* d_conc, const int nx, const int ny)
{
	hipMemcpy(conc[0], d_conc, nx * ny * sizeof(fp_t),
               hipMemcpyDeviceToHost);
}
