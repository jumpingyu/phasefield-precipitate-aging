#include "hip/hip_runtime.h"
/******************************************************************************
 *                       Code generated with sympy 1.4                        *
 *                                                                            *
 *              See http://www.sympy.org/ for more information.               *
 *                                                                            *
 *                  This file is part of 'PrecipitateAging'                   *
 ******************************************************************************/
#include "parabola625.cuh"
#include <math.h>

__device__ double d_h(double x)
{

	double h_result;
	h_result = x*x*x*(6.0*x*x - 15.0*x + 10.0);
	return h_result;

}

__device__ double d_hprime(double x)
{

	double hprime_result;
	hprime_result = 30.0*x*x*(1.0 - x)*(1.0 - x);
	return hprime_result;

}

__device__ double d_interface_profile(double z)
{

	double interface_profile_result;
	interface_profile_result = 1.0/2.0 - 1.0/2.0*tanh(z);
	return interface_profile_result;

}

__device__ double d_kT()
{

	double kT_result;
	kT_result = 1.5782889043500002e-20;
	return kT_result;

}

__device__ double d_RT()
{

	double RT_result;
	RT_result = 9504.6840941999999;
	return RT_result;

}

__device__ double d_Vm()
{

	double Vm_result;
	Vm_result = 1.0000000000000001e-5;
	return Vm_result;

}

__device__ double d_xe_gam_Cr()
{

	double xe_gam_Cr_result;
	xe_gam_Cr_result = 0.55855269488699388;
	return xe_gam_Cr_result;

}

__device__ double d_xe_gam_Nb()
{

	double xe_gam_Nb_result;
	xe_gam_Nb_result = 0.010717747618334031;
	return xe_gam_Nb_result;

}

__device__ double d_xe_del_Cr()
{

	double xe_del_Cr_result;
	xe_del_Cr_result = 0.031881757028651196;
	return xe_del_Cr_result;

}

__device__ double d_xe_del_Nb()
{

	double xe_del_Nb_result;
	xe_del_Nb_result = 0.16874796371854489;
	return xe_del_Nb_result;

}

__device__ double d_xe_lav_Cr()
{

	double xe_lav_Cr_result;
	xe_lav_Cr_result = 0.44421217837672827;
	return xe_lav_Cr_result;

}

__device__ double d_xe_lav_Nb()
{

	double xe_lav_Nb_result;
	xe_lav_Nb_result = 0.17170586512707406;
	return xe_lav_Nb_result;

}

__device__ double d_matrix_min_Cr()
{

	double matrix_min_Cr_result;
	matrix_min_Cr_result = 0.27939999999999998;
	return matrix_min_Cr_result;

}

__device__ double d_matrix_max_Cr()
{

	double matrix_max_Cr_result;
	matrix_max_Cr_result = 0.32879999999999998;
	return matrix_max_Cr_result;

}

__device__ double d_matrix_min_Nb()
{

	double matrix_min_Nb_result;
	matrix_min_Nb_result = 0.021499999999999998;
	return matrix_min_Nb_result;

}

__device__ double d_matrix_max_Nb()
{

	double matrix_max_Nb_result;
	matrix_max_Nb_result = 0.0269;
	return matrix_max_Nb_result;

}

__device__ double d_enrich_min_Cr()
{

	double enrich_min_Cr_result;
	enrich_min_Cr_result = 0.24729999999999999;
	return enrich_min_Cr_result;

}

__device__ double d_enrich_max_Cr()
{

	double enrich_max_Cr_result;
	enrich_max_Cr_result = 0.29670000000000002;
	return enrich_max_Cr_result;

}

__device__ double d_enrich_min_Nb()
{

	double enrich_min_Nb_result;
	enrich_min_Nb_result = 0.16589999999999999;
	return enrich_min_Nb_result;

}

__device__ double d_enrich_max_Nb()
{

	double enrich_max_Nb_result;
	enrich_max_Nb_result = 0.1726;
	return enrich_max_Nb_result;

}

__device__ double d_xr_gam_Cr(double P_del, double P_lav)
{

	double xr_gam_Cr_result;
	xr_gam_Cr_result = -9.6984079989082258e-10*P_del + 6.3118903213767551e-10*P_lav + 0.55855269488699388;
	return xr_gam_Cr_result;

}

__device__ double d_xr_gam_Nb(double P_del, double P_lav)
{

	double xr_gam_Nb_result;
	xr_gam_Nb_result = 1.0048047396144418e-10*P_del - 1.1106547204101917e-11*P_lav + 0.010717747618334031;
	return xr_gam_Nb_result;

}

__device__ double d_xr_del_Cr(double P_del, double P_lav)
{

	double xr_del_Cr_result;
	xr_del_Cr_result = -6.6194274436412276e-11*P_del - 1.5374687132506466e-10*P_lav + 0.031881757028651196;
	return xr_del_Cr_result;

}

__device__ double d_xr_del_Nb(double P_del, double P_lav)
{

	double xr_del_Nb_result;
	xr_del_Nb_result = -1.9581600075232547e-11*P_del + 3.9815435461767849e-10*P_lav + 0.16874796371854489;
	return xr_del_Nb_result;

}

__device__ double d_xr_lav_Cr(double P_del, double P_lav)
{

	double xr_lav_Cr_result;
	xr_lav_Cr_result = -1.2953044143338369e-9*P_del + 4.3416027339639082e-10*P_lav + 0.44421217837672827;
	return xr_lav_Cr_result;

}

__device__ double d_xr_lav_Nb(double P_del, double P_lav)
{

	double xr_lav_Nb_result;
	xr_lav_Nb_result = 2.6966712244807525e-10*P_del - 6.3173100035954083e-11*P_lav + 0.17170586512707406;
	return xr_lav_Nb_result;

}

__device__ double d_inv_fict_det(double f_del, double f_gam, double f_lav)
{

	double inv_fict_det_result;
	inv_fict_det_result = 10.511929918754895/(0.063619913592430452*f_del*f_del + 0.77472418933378007*f_del*f_gam + 1.0*f_del*f_lav + 0.11406419308318602*f_gam*f_gam + 0.29936120577567832*f_gam*f_lav + 0.076367679948657952*f_lav*f_lav);
	return inv_fict_det_result;

}

__device__ double d_fict_gam_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_gam_Cr_result;
	fict_gam_Cr_result = 0.081719161132759205*INV_DET*(0.94267830414080844*XCR*f_del + 0.13278315265091145*XCR*f_gam + 0.27475106182917869*XCR*f_lav + 0.48142139214471374*XNB*f_del + 1.0*XNB*f_lav - 0.069926420929843164*f_del*f_del - 0.03219039130480722*f_del*f_gam - 0.028703356204519043*f_del*f_lav - 0.028515077359655715*f_gam*f_lav - 0.24409812910594283*f_lav*f_lav);
	return fict_gam_Cr_result;

}

__device__ double d_fict_gam_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_gam_Nb_result;
	fict_gam_Nb_result = -0.010850927847195595*INV_DET*(0.75572285002798056*XCR*f_del - 0.063676254673428662*XCR*f_lav + 0.30738018673156009*XNB*f_del - 1.0*XNB*f_gam - 0.55532654346045196*XNB*f_lav - 0.081941433925801291*f_del*f_del - 0.32945245368491854*f_del*f_gam - 0.38670236006564801*f_del*f_lav + 0.18509555516266776*f_gam*f_lav + 0.11646290035390382*f_lav*f_lav);
	return fict_gam_Nb_result;

}

__device__ double d_fict_del_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_del_Cr_result;
	fict_del_Cr_result = 0.046966148206032843*INV_DET*(0.12886223924609808*XCR*f_del - 0.07101626075977073*XCR*f_gam - 0.20028342359736442*XCR*f_lav - 0.83765336993033834*XNB*f_gam - 1.0*XNB*f_lav + 0.12166908885811056*f_del*f_gam + 0.18246772627205363*f_del*f_lav + 0.056009953433358468*f_gam*f_gam + 0.31729459226818346*f_gam*f_lav + 0.26560576240348166*f_lav*f_lav);
	return fict_del_Cr_result;

}

__device__ double d_fict_del_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_del_Nb_result;
	fict_del_Nb_result = 0.10453655111851477*INV_DET*(0.078444276479274974*XCR*f_gam + 0.20923881653433052*XCR*f_lav + 0.057895185577072183*XNB*f_del + 0.73691813445330934*XNB*f_gam + 1.0*XNB*f_lav - 0.0085055473679878923*f_del*f_gam - 0.031796338023487768*f_del*f_lav - 0.034197271344486177*f_gam*f_gam - 0.24299677574697207*f_gam*f_lav - 0.25292500971253778*f_lav*f_lav);
	return fict_del_Nb_result;

}

__device__ double d_fict_lav_Cr(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_lav_Cr_result;
	fict_lav_Cr_result = 0.10453655111851476*INV_DET*(1.0*XCR*f_del + 0.057643074984273575*XCR*f_gam + 0.0694958662006743*XCR*f_lav + 0.44927967972452598*XNB*f_del - 0.78172811574884349*XNB*f_gam - 0.081979041619570731*f_del*f_del - 0.12011579223114696*f_del*f_gam - 0.11933127186562477*f_del*f_lav + 0.022291037694796172*f_gam*f_gam + 0.19081837052380662*f_gam*f_lav);
	return fict_lav_Cr_result;

}

__device__ double d_fict_lav_Nb(double INV_DET, double XCR, double XNB, double f_del, double f_gam, double f_lav)
{

	double fict_lav_Nb_result;
	fict_lav_Nb_result = -0.02643990820696555*INV_DET*(0.82727610358556924*XCR*f_del + 0.026132709676314116*XCR*f_gam + 0.35577056025509707*XNB*f_del - 0.84918699858043711*XNB*f_gam - 0.27476866079657325*XNB*f_lav - 0.12571448770378987*f_del*f_del - 0.80204383828035919*f_del*f_gam - 1.0*f_del*f_lav - 0.075963142465736502*f_gam*f_gam - 0.047796328138628434*f_gam*f_lav);
	return fict_lav_Nb_result;

}

__device__ double d_s_delta()
{

	double s_delta_result;
	s_delta_result = 0.13;
	return s_delta_result;

}

__device__ double d_s_laves()
{

	double s_laves_result;
	s_laves_result = 0.13;
	return s_laves_result;

}

__device__ double d_g_gam(double XCR, double XNB)
{

	double g_gam_result;
	g_gam_result = 2005672715.5837781*pow(XCR - 0.55855269488699388, 2) + (14703563128.545319*XCR - 8212714809.8900261)*(XNB - 0.010717747618334031) + 62431733279.319511*pow(XNB - 0.010717747618334031, 2);
	return g_gam_result;

}

__device__ double d_g_del(double XCR, double XNB)
{

	double g_del_result;
	g_del_result = 15567985511.489613*pow(XCR - 0.031881757028651196, 2) + (17972099186.595295*XCR - 572982099.56185102)*(XNB - 0.16874796371854489) + 13383100915.719385*pow(XNB - 0.16874796371854489, 2);
	return g_del_result;

}

__device__ double d_g_lav(double XCR, double XNB)
{

	double g_lav_result;
	g_lav_result = 6897850581.6836567*pow(XCR - 0.44421217837672827, 2) + (57317655210.986305*XCR - 25461200480.718456)*(XNB - 0.17170586512707406) + 134480681274.87074*pow(XNB - 0.17170586512707406, 2);
	return g_lav_result;

}

__device__ double d_dg_gam_dxCr(double XCR, double XNB)
{

	double dg_gam_dxCr_result;
	dg_gam_dxCr_result = 4011345431.1675563*XCR + 14703563128.545319*XNB - 2398136879.4032598;
	return dg_gam_dxCr_result;

}

__device__ double d_dg_gam_dxNb(double XCR, double XNB)
{

	double dg_gam_dxNb_result;
	dg_gam_dxNb_result = 14703563128.545319*XCR + 124863466558.63902*XNB - 9550969931.2158108;
	return dg_gam_dxNb_result;

}

__device__ double d_dg_del_dxCr(double XCR, double XNB)
{

	double dg_del_dxCr_result;
	dg_del_dxCr_result = 31135971022.979225*XCR + 17972099186.595295*XNB - 4025424604.4914207;
	return dg_del_dxCr_result;

}

__device__ double d_dg_del_dxNb(double XCR, double XNB)
{

	double dg_del_dxNb_result;
	dg_del_dxNb_result = 17972099186.595295*XCR + 26766201831.43877*XNB - 5089724155.0967312;
	return dg_del_dxNb_result;

}

__device__ double d_dg_lav_dxCr(double XCR, double XNB)
{

	double dg_lav_dxCr_result;
	dg_lav_dxCr_result = 13795701163.367313*XCR + 57317655210.986305*XNB - 15969996041.071507;
	return dg_lav_dxCr_result;

}

__device__ double d_dg_lav_dxNb(double XCR, double XNB)
{

	double dg_lav_dxNb_result;
	dg_lav_dxNb_result = 57317655210.986305*XCR + 268961362549.74149*XNB - 71643443923.07843;
	return dg_lav_dxNb_result;

}

__device__ double d_d2g_gam_dxCrCr()
{

	double d2g_gam_dxCrCr_result;
	d2g_gam_dxCrCr_result = 4011345431.1675563;
	return d2g_gam_dxCrCr_result;

}

__device__ double d_d2g_gam_dxCrNb()
{

	double d2g_gam_dxCrNb_result;
	d2g_gam_dxCrNb_result = 14703563128.545319;
	return d2g_gam_dxCrNb_result;

}

__device__ double d_d2g_gam_dxNbCr()
{

	double d2g_gam_dxNbCr_result;
	d2g_gam_dxNbCr_result = 14703563128.545319;
	return d2g_gam_dxNbCr_result;

}

__device__ double d_d2g_gam_dxNbNb()
{

	double d2g_gam_dxNbNb_result;
	d2g_gam_dxNbNb_result = 124863466558.63902;
	return d2g_gam_dxNbNb_result;

}

__device__ double d_d2g_del_dxCrCr()
{

	double d2g_del_dxCrCr_result;
	d2g_del_dxCrCr_result = 31135971022.979225;
	return d2g_del_dxCrCr_result;

}

__device__ double d_d2g_del_dxCrNb()
{

	double d2g_del_dxCrNb_result;
	d2g_del_dxCrNb_result = 17972099186.595295;
	return d2g_del_dxCrNb_result;

}

__device__ double d_d2g_del_dxNbCr()
{

	double d2g_del_dxNbCr_result;
	d2g_del_dxNbCr_result = 17972099186.595295;
	return d2g_del_dxNbCr_result;

}

__device__ double d_d2g_del_dxNbNb()
{

	double d2g_del_dxNbNb_result;
	d2g_del_dxNbNb_result = 26766201831.43877;
	return d2g_del_dxNbNb_result;

}

__device__ double d_d2g_lav_dxCrCr()
{

	double d2g_lav_dxCrCr_result;
	d2g_lav_dxCrCr_result = 13795701163.367313;
	return d2g_lav_dxCrCr_result;

}

__device__ double d_d2g_lav_dxCrNb()
{

	double d2g_lav_dxCrNb_result;
	d2g_lav_dxCrNb_result = 57317655210.986305;
	return d2g_lav_dxCrNb_result;

}

__device__ double d_d2g_lav_dxNbCr()
{

	double d2g_lav_dxNbCr_result;
	d2g_lav_dxNbCr_result = 57317655210.986305;
	return d2g_lav_dxNbCr_result;

}

__device__ double d_d2g_lav_dxNbNb()
{

	double d2g_lav_dxNbNb_result;
	d2g_lav_dxNbNb_result = 268961362549.74149;
	return d2g_lav_dxNbNb_result;

}

__device__ double d_D_CrCr(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_CrCr_result;
	D_CrCr_result = 1.0000000000000003e-15*(3268536058.0499763*phi_del + 3268536058.0499763*phi_lav + 14703563128.545319)*(XCR*XNB*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) - XCR*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB) + XNB*(1 - XCR)*(-8.2216387898155807e-8*XCR*(-XCR - XNB + 1) - 9.6755272489124536e-20*XCR - 3.5027570743586952e-21)) + 1.0000000000000003e-15*(27124625591.811668*phi_del + 27124625591.811668*phi_lav + 4011345431.1675563)*(pow(XCR, 2)*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB) + pow(XCR, 2)*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) + (1 - XCR)*(1 - XCR)*(8.2216387898155807e-8*XCR*(-XCR - XNB + 1) + 9.6755272489124536e-20*XCR + 3.5027570743586952e-21));
	return D_CrCr_result;

}

__device__ double d_D_CrNb(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_CrNb_result;
	D_CrNb_result = 1.0000000000000003e-15*(-98097264727.200256*phi_del - 98097264727.200256*phi_lav + 124863466558.63902)*(XCR*XNB*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) - XCR*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB) + XNB*(1 - XCR)*(-8.2216387898155807e-8*XCR*(-XCR - XNB + 1) - 9.6755272489124536e-20*XCR - 3.5027570743586952e-21)) + 1.0000000000000003e-15*(3268536058.0499763*phi_del + 3268536058.0499763*phi_lav + 14703563128.545319)*(pow(XCR, 2)*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB) + pow(XCR, 2)*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) + (1 - XCR)*(1 - XCR)*(8.2216387898155807e-8*XCR*(-XCR - XNB + 1) + 9.6755272489124536e-20*XCR + 3.5027570743586952e-21));
	return D_CrNb_result;

}

__device__ double d_D_NbCr(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_NbCr_result;
	D_NbCr_result = 1.0000000000000003e-15*(3268536058.0499763*phi_del + 3268536058.0499763*phi_lav + 14703563128.545319)*(pow(XNB, 2)*(8.2216387898155807e-8*XCR*(-XCR - XNB + 1) + 9.6755272489124536e-20*XCR + 3.5027570743586952e-21) + pow(XNB, 2)*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) + (1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB)*(1 - XNB)) + 1.0000000000000003e-15*(27124625591.811668*phi_del + 27124625591.811668*phi_lav + 4011345431.1675563)*(XCR*XNB*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) - XCR*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB) + XNB*(1 - XCR)*(-8.2216387898155807e-8*XCR*(-XCR - XNB + 1) - 9.6755272489124536e-20*XCR - 3.5027570743586952e-21));
	return D_NbCr_result;

}

__device__ double d_D_NbNb(double XCR, double XNB, double phi_del, double phi_lav)
{

	double D_NbNb_result;
	D_NbNb_result = 1.0000000000000003e-15*(-98097264727.200256*phi_del - 98097264727.200256*phi_lav + 124863466558.63902)*(pow(XNB, 2)*(8.2216387898155807e-8*XCR*(-XCR - XNB + 1) + 9.6755272489124536e-20*XCR + 3.5027570743586952e-21) + pow(XNB, 2)*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) + (1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB)*(1 - XNB)) + 1.0000000000000003e-15*(3268536058.0499763*phi_del + 3268536058.0499763*phi_lav + 14703563128.545319)*(XCR*XNB*(2.0938866959006431e-8*XCR*(-XCR - XNB + 1) + 9.8428461923389931e-20*XCR - 1.0199962450633582e-21*XNB + 1.8295676400933012e-21) - XCR*(1.7235555733323437e-20 - 1.4581024012583029e-20*XNB)*(1 - XNB) + XNB*(1 - XCR)*(-8.2216387898155807e-8*XCR*(-XCR - XNB + 1) - 9.6755272489124536e-20*XCR - 3.5027570743586952e-21));
	return D_NbNb_result;

}
