#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "cuda_data.h"
#include "numerics.h"
#include "mesh.h"

#include "cuda_kernels.cuh"
#include "parabola625.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new,
                                   const int nx, const int ny, const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm/2;
	const int src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)], NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_conc_new[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__global__ void composition_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                   fp_t* d_phi_del_old,
                                   fp_t* d_phi_lav_old,
                                   fp_t* d_gam_Cr_old,  fp_t* d_gam_Nb_old,
                                   fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                   fp_t* d_gam_Cr_new,  fp_t* d_gam_Nb_new,
                                   const int nx, const int ny, const int nm,
                                   const fp_t D_CrCr, const fp_t D_CrNb,
                                   const fp_t D_NbCr, const fp_t D_NbNb,
                                   const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

        /* Cahn-Hilliard equations of motion for composition */
        const fp_t lap_mu_Cr = D_CrCr * d_gam_Cr_new[idx]
                             + D_NbCr * d_gam_Nb_new[idx];
        const fp_t lap_mu_Nb = D_CrNb * d_gam_Cr_new[idx]
                             + D_NbNb * d_gam_Nb_new[idx];

        d_conc_Cr_new[idx] = d_conc_Cr_old[idx] + dt * lap_mu_Cr;
        d_conc_Nb_new[idx] = d_conc_Nb_old[idx] + dt * lap_mu_Nb;
    }
	__syncthreads();
}

__global__ void delta_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                             fp_t* d_phi_del_old,
                             fp_t* d_phi_lav_old,
                             fp_t* d_gam_Cr_old,  fp_t* d_gam_Nb_old,
                             fp_t* d_phi_del_new,
                             const int nx, const int ny, const int nm,
                             const fp_t alpha, const fp_t kappa, const fp_t omega,
                             const fp_t M_del, const fp_t M_lav,
                             const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
        const fp_t del_Cr = d_fict_del_Cr(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], f_del, 1.-f_del-f_lav, f_lav);
        const fp_t del_Nb = d_fict_del_Nb(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], f_del, 1.-f_del-f_lav, f_lav);
        const fp_t del_nrg = d_g_del(del_Cr, del_Nb);

        /* pressure */
        const fp_t P_del = gam_nrg - del_nrg - dgGdxCr * (d_gam_Cr_old[idx] - del_Cr) - dgGdxNb * (d_gam_Nb_old[idx] - del_Nb);

        /* variational derivative */
        const fp_t dFdPhi_del = -d_hprime(d_phi_del_old[idx]) * P_del
          + 2. * omega * d_phi_del_old[idx] * (d_phi_del_old[idx] - 1.) * (2. * d_phi_del_old[idx] - 1.)
          + 2. * alpha * d_phi_del_old[idx] * (d_phi_lav_old[idx] * d_phi_lav_old[idx])
          - kappa * d_phi_del_new[idx];

        /* Allen-Cahn equation of motion for delta phase */
        d_phi_del_new[idx] = d_phi_del_old[idx] - dt * M_del * dFdPhi_del;
    }
	__syncthreads();
}

__global__ void laves_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                             fp_t* d_phi_del_old,
                             fp_t* d_phi_lav_old,
                             fp_t* d_gam_Cr_old,  fp_t* d_gam_Nb_old,
                             fp_t* d_phi_lav_new,
                             const int nx, const int ny, const int nm,
                             const fp_t alpha, const fp_t kappa, const fp_t omega,
                             const fp_t M_del, const fp_t M_lav,
                             const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
        const fp_t lav_Cr = d_fict_lav_Cr(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], f_lav, 1.-f_del-f_lav, f_lav);
        const fp_t lav_Nb = d_fict_lav_Nb(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], f_lav, 1.-f_del-f_lav, f_lav);
        const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

        /* pressure */
        const fp_t P_lav = gam_nrg - lav_nrg - dgGdxCr * (d_gam_Cr_old[idx] - lav_Cr) - dgGdxNb * (d_gam_Nb_old[idx] - lav_Nb);

        /* variational derivative */
        const fp_t dFdPhi_lav = -d_hprime(d_phi_lav_old[idx]) * P_lav
          + 2. * omega * d_phi_lav_old[idx] * (d_phi_lav_old[idx] - 1.) * (2. * d_phi_lav_old[idx] - 1.)
          + 2. * alpha * d_phi_lav_old[idx] * (d_phi_del_old[idx] * d_phi_del_old[idx])
          - kappa * d_phi_lav_new[idx];

        /* Allen-Cahn equation of motion for delta phase */
        d_phi_lav_new[idx] = d_phi_lav_old[idx] - dt * M_lav * dFdPhi_lav;
    }
	__syncthreads();
}

__global__ void fictitious_kernel(fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                  fp_t* d_phi_del_new,
                                  fp_t* d_phi_lav_new,
                                  fp_t* d_gam_Cr_new,  fp_t* d_gam_Nb_new,
                                  const int nx, const int ny, const int nm)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_new[idx]);
		const fp_t f_lav = d_h(d_phi_lav_new[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

        d_gam_Cr_new[idx] = d_fict_gam_Cr(inv_fict_det, d_conc_Cr_new[idx], d_conc_Nb_new[idx],
                                          f_del, 1.-f_del-f_lav, f_lav);
        d_gam_Nb_new[idx] = d_fict_gam_Nb(inv_fict_det, d_conc_Cr_new[idx], d_conc_Nb_new[idx],
                                          f_del, 1.-f_del-f_lav, f_lav);
    }
	__syncthreads();
}

void device_boundaries(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old,
	    dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    nx, ny, nm
	);
}

void device_laplacian_boundaries(struct CudaData* dev,
                            const int nx, const int ny, const int nm,
                            const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new,
	    dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm
	);
}

void device_laplacian(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Cr_old, dev->conc_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Nb_old, dev->conc_Nb_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Cr_old, dev->gam_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Nb_old, dev->gam_Nb_new, nx, ny, nm);
}

void device_evolution(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t D_CrCr, const fp_t D_CrNb,
                      const fp_t D_NbCr, const fp_t D_NbNb,
                      const fp_t alpha, const fp_t kappa, const fp_t omega,
                      const fp_t M_del, const fp_t M_lav,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	composition_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr_old,  dev->gam_Nb_old,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->gam_Cr_new,  dev->gam_Nb_new,
	    nx, ny, nm,
	    D_CrCr, D_CrNb,
	    D_NbCr, D_NbNb,
	    dt);
	delta_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    dev->phi_del_new,
	    nx, ny, nm,
	    alpha, kappa, omega,
	    M_del, M_lav,
	    dt);
	laves_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    dev->phi_lav_new,
	    nx, ny, nm,
	    alpha, kappa, omega,
	    M_del, M_lav,
	    dt);
	fictitious_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm);
}

void read_out_result(struct CudaData* dev, struct HostData* host, const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Cr_new[0], dev->gam_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Nb_new[0], dev->gam_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}
