/**
 \file  data.cu
 \brief Implementation of functions to create and destroy CudaData struct
*/

#include <hiprand.h>
#include "data.cuh"
#include "discretization.cuh"

void init_cuda(struct HostData* host,
               const int nx, const int ny, const int nm,
			   const fp_t* kappa, const fp_t* omega, const fp_t* Lmob, 
			   struct CudaData* dev)
{
	/* allocate memory on device */
	hipMalloc((void**) &(dev->conc_Cr_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Cr_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Ni), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->phi_del_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_del_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi),         nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->prng), nx * ny * sizeof(hiprandState));

	/* transfer mask to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask), host->mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer mobility data to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_Kapp), kappa, NP * sizeof(fp_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_Omeg), omega, NP * sizeof(fp_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_Lmob), Lmob,  NP * sizeof(fp_t));

	/* transfer data from host in to GPU */
	hipMemcpy(dev->conc_Cr_old, host->conc_Cr_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->conc_Nb_old, host->conc_Nb_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);

	hipMemcpy(dev->phi_del_old, host->phi_del_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->phi_lav_old, host->phi_lav_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
}

void free_cuda(struct CudaData* dev)
{
	/* free memory on device */
	hipFree(dev->conc_Cr_old);
	hipFree(dev->conc_Cr_new);
	hipFree(dev->conc_Nb_old);
	hipFree(dev->conc_Nb_new);
	hipFree(dev->conc_Ni);

	hipFree(dev->phi_del_old);
	hipFree(dev->phi_del_new);
	hipFree(dev->phi_lav_old);
	hipFree(dev->phi_lav_new);
	hipFree(dev->phi);

	hipFree(dev->prng);
}

void read_out_result(struct CudaData* dev, struct HostData* host,
                     const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}
