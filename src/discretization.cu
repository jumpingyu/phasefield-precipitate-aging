#include "hip/hip_runtime.h"
/**
 \file  discretization.cu
 \brief Implementation of discretized equations with CUDA acceleration
 Contains functions for boundary conditions, equations of motion, and nucleation.
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>

#include "data.cuh"
#include "discretization.cuh"
#include "parabola625.cuh"
#include "nucleation.cuh"

/**
 \brief Convenience function for checking CUDA runtime API results
 No-op in release builds.
*/
hipError_t checkCuda(hipError_t result)
{
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

// Convolution mask array on the GPU, allocated in protected memory
__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

// Kinetic parameter arrays on the GPU, allocated in protected memory
__constant__ fp_t d_Kapp[NP];
__constant__ fp_t d_Omeg[NP];
__constant__ fp_t d_Lmob[NP];

float nTiles(int domain_size, int tile_loc, int mask_size)
{
	return ceil(float(domain_size) / float(tile_loc - mask_size + 1));
}

__global__ void boundary_kernel(fp_t* d_conc_Cr,
								fp_t* d_conc_Nb,
                                fp_t* d_phi_del,
								fp_t* d_phi_lav,
                                const int nx,
                                const int ny,
                                const int nm)
{
	/* determine indices on which to operate */
	const int row = blockDim.y * blockIdx.y + threadIdx.y;
	const int col = blockDim.x * blockIdx.x + threadIdx.x;

	/* apply no-flux boundary conditions: inside to out, sequence matters */

	for (int offset = 0; offset < nm / 2; offset++) {
		const int ilo = nm / 2 - offset;
		const int ihi = nx - 1 - nm / 2 + offset;
		const int jlo = nm / 2 - offset;
		const int jhi = ny - 1 - nm / 2 + offset;

		if (ilo - 1 == col && row < ny) {
			/* left condition */
			d_conc_Cr[row * nx + ilo - 1] = d_conc_Cr[row * nx + ilo];
			d_conc_Nb[row * nx + ilo - 1] = d_conc_Nb[row * nx + ilo];
			d_phi_del[row * nx + ilo - 1] = d_phi_del[row * nx + ilo];
			d_phi_lav[row * nx + ilo - 1] = d_phi_lav[row * nx + ilo];
		}
		if (ihi + 1 == col && row < ny) {
			/* right condition */
			d_conc_Cr[row * nx + ihi + 1] = d_conc_Cr[row * nx + ihi];
			d_conc_Nb[row * nx + ihi + 1] = d_conc_Nb[row * nx + ihi];
			d_phi_del[row * nx + ihi + 1] = d_phi_del[row * nx + ihi];
			d_phi_lav[row * nx + ihi + 1] = d_phi_lav[row * nx + ihi];
		}
		if (jlo - 1 == row && col < nx) {
			/* bottom condition */
			d_conc_Cr[(jlo - 1) * nx + col] = d_conc_Cr[jlo * nx + col];
			d_conc_Nb[(jlo - 1) * nx + col] = d_conc_Nb[jlo * nx + col];
			d_phi_del[(jlo - 1) * nx + col] = d_phi_del[jlo * nx + col];
			d_phi_lav[(jlo - 1) * nx + col] = d_phi_lav[jlo * nx + col];
		}
		if (jhi + 1 == row && col < nx) {
			/* top condition */
			d_conc_Cr[(jhi + 1) * nx + col] = d_conc_Cr[jhi * nx + col];
			d_conc_Nb[(jhi + 1) * nx + col] = d_conc_Nb[jhi * nx + col];
			d_phi_del[(jhi + 1) * nx + col] = d_phi_del[jhi * nx + col];
			d_phi_lav[(jhi + 1) * nx + col] = d_phi_lav[jhi * nx + col];
		}
	}
}

void device_boundaries(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    nx, ny, nm);
}

void device_laplacian_boundaries(struct CudaData* dev,
                                 const int nx, const int ny, const int nm,
                                 const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    nx, ny, nm);
}

__global__ void convolution_kernel(fp_t* d_old,
                                   fp_t* d_new,
                                   const int nx,
								   const int ny,
								   const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm / 2;
	const int src_y = dst_y - nm / 2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ double4 d_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		(d_tile[til_nx * til_y + til_x]).x = d_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				const double4& mid = (d_tile[til_nx * (til_y + j) + (til_x + i)]);
				value += d_mask[j * nm + i] * mid.x;
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)],
		         NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_new[nx * dst_y + dst_x] = value;
		}
	}
}

__device__ fp_t discrete_laplacian(const fp_t& D_middle,
								   const fp_t& D_left, const fp_t& D_right,
								   const fp_t& D_bottom, const fp_t& D_top,
								   const fp_t& c_middle,
								   const fp_t& c_left, const fp_t& c_right,
								   const fp_t& c_bottom, const fp_t& c_top,
								   const fp_t& dx, const fp_t& dy)
{
	// Five-point stencil
	return ( (D_right + D_middle) * (c_right - c_middle)
	       - (D_middle + D_left) * (c_middle - c_left)) / (2.0 * dx * dx)
	     + ( (D_top + D_middle) * (c_top - c_middle)
		   - (D_middle + D_bottom) * (c_middle - c_bottom)) / (2.0 * dy * dy);
}

__device__ void fictitious(const fp_t xCr, const fp_t xNb, const fp_t pDel, const fp_t pLav,
                           fp_t* gam_Cr, fp_t* gam_Nb, fp_t* del_Cr, fp_t* del_Nb, fp_t* lav_Cr, fp_t* lav_Nb)
{
	const fp_t pGam = 1.0 - pDel - pLav;
    const fp_t inv_det = d_inv_fict_det(pDel, pGam, pLav);
    *gam_Cr = d_fict_gam_Cr(inv_det, xCr, xNb, pDel, pGam, pLav);
    *del_Cr = d_fict_del_Cr(inv_det, xCr, xNb, pDel, pGam, pLav);
    *lav_Cr = d_fict_lav_Cr(inv_det, xCr, xNb, pDel, pGam, pLav);
    *gam_Nb = d_fict_gam_Nb(inv_det, xCr, xNb, pDel, pGam, pLav);
    *del_Nb = d_fict_del_Nb(inv_det, xCr, xNb, pDel, pGam, pLav);
    *lav_Nb = d_fict_lav_Nb(inv_det, xCr, xNb, pDel, pGam, pLav);
}

__device__ fp_t D_gam_CrCr(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_gam_dxCrCr() + d_M_CrNb(xCr, xNb) * d_d2g_gam_dxCrNb();
}
__device__ fp_t D_gam_CrNb(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_gam_dxNbCr() + d_M_CrNb(xCr, xNb) * d_d2g_gam_dxNbNb();
}
__device__ fp_t D_gam_NbCr(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_gam_dxCrCr() + d_M_NbNb(xCr, xNb) * d_d2g_gam_dxCrNb();
}
__device__ fp_t D_gam_NbNb(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_gam_dxNbCr() + d_M_NbNb(xCr, xNb) * d_d2g_gam_dxNbNb();
}

__device__ fp_t D_del_CrCr(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_del_dxCrCr() + d_M_CrNb(xCr, xNb) * d_d2g_del_dxCrNb();
}
__device__ fp_t D_del_CrNb(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_del_dxNbCr() + d_M_CrNb(xCr, xNb) * d_d2g_del_dxNbNb();
}
__device__ fp_t D_del_NbCr(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_del_dxCrCr() + d_M_NbNb(xCr, xNb) * d_d2g_del_dxCrNb();
}
__device__ fp_t D_del_NbNb(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_del_dxNbCr() + d_M_NbNb(xCr, xNb) * d_d2g_del_dxNbNb();
}

__device__ fp_t D_lav_CrCr(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_lav_dxCrCr() + d_M_CrNb(xCr, xNb) * d_d2g_lav_dxCrNb();
}
__device__ fp_t D_lav_CrNb(const fp_t xCr, const fp_t xNb) {
	return d_M_CrCr(xCr, xNb) * d_d2g_lav_dxNbCr() + d_M_CrNb(xCr, xNb) * d_d2g_lav_dxNbNb();
}
__device__ fp_t D_lav_NbCr(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_lav_dxCrCr() + d_M_NbNb(xCr, xNb) * d_d2g_lav_dxCrNb();
}
__device__ fp_t D_lav_NbNb(const fp_t xCr, const fp_t xNb) {
	return d_M_NbCr(xCr, xNb) * d_d2g_lav_dxNbCr() + d_M_NbNb(xCr, xNb) * d_d2g_lav_dxNbNb();
}

__global__ void chemical_convolution_kernel(fp_t* d_phi_del_old, fp_t* d_phi_lav_old,
        fp_t* d_conc_Cr_old, fp_t* d_conc_Cr_new,
        fp_t* d_conc_Nb_old, fp_t* d_conc_Nb_new,
        const int nx, const int ny, const int nm,
        const fp_t dx, const fp_t dy)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm / 2;
	const int src_y = dst_y - nm / 2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ double4 d_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		(d_tile[til_nx * til_y + til_x]).x = d_conc_Cr_old[nx * src_y + src_x];
		(d_tile[til_nx * til_y + til_x]).y = d_conc_Nb_old[nx * src_y + src_x];
		(d_tile[til_nx * til_y + til_x]).z = d_phi_del_old[nx * src_y + src_x];
		(d_tile[til_nx * til_y + til_x]).w = d_phi_lav_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the 5-point Laplacian with variable coefficients */
	if (til_x < dst_nx && til_y < dst_ny) {
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)] */
		const size_t til_mdx = til_x + 1;
		const size_t til_mdy = til_y + 1;
		const size_t til_lft = til_x;
		const size_t til_rgt = til_x + 2;
		const size_t til_bot = til_y;
		const size_t til_top = til_y + 2;

		const double4* mid = &(d_tile[til_nx * til_mdy + til_mdx]);
		const double4* lft = &(d_tile[til_nx * til_mdy + til_lft]);
		const double4* rgt = &(d_tile[til_nx * til_mdy + til_rgt]);
		const double4* bot = &(d_tile[til_nx * til_bot + til_mdx]);
		const double4* top = &(d_tile[til_nx * til_top + til_mdx]);

		const fp_t mid_pDel = d_p(mid->z);
		const fp_t mid_pLav = d_p(mid->w);
		const fp_t mid_pGam = 1.0 - mid_pDel - mid_pLav;

		const fp_t lft_pDel = d_p(lft->z);
		const fp_t lft_pLav = d_p(lft->w);
		const fp_t lft_pGam = 1.0 - lft_pDel - lft_pLav;

		const fp_t rgt_pDel = d_p(rgt->z);
		const fp_t rgt_pLav = d_p(rgt->w);
		const fp_t rgt_pGam = 1.0 - rgt_pDel - rgt_pLav;

		const fp_t bot_pDel = d_p(bot->z);
		const fp_t bot_pLav = d_p(bot->w);
		const fp_t bot_pGam = 1.0 - bot_pDel - bot_pLav;

		const fp_t top_pDel = d_p(top->z);
		const fp_t top_pLav = d_p(top->w);
		const fp_t top_pGam = 1.0 - top_pDel - top_pLav;

		// Fictitious compositions
		fp_t mid_gam_Cr, mid_gam_Nb, mid_del_Cr, mid_del_Nb, mid_lav_Cr, mid_lav_Nb;
		fp_t lft_gam_Cr, lft_gam_Nb, lft_del_Cr, lft_del_Nb, lft_lav_Cr, lft_lav_Nb;
		fp_t rgt_gam_Cr, rgt_gam_Nb, rgt_del_Cr, rgt_del_Nb, rgt_lav_Cr, rgt_lav_Nb;
		fp_t bot_gam_Cr, bot_gam_Nb, bot_del_Cr, bot_del_Nb, bot_lav_Cr, bot_lav_Nb;
		fp_t top_gam_Cr, top_gam_Nb, top_del_Cr, top_del_Nb, top_lav_Cr, top_lav_Nb;
        fictitious(mid->x, mid->y, mid_pDel, mid_pLav, &mid_gam_Cr, &mid_gam_Nb, &mid_del_Cr, &mid_del_Nb, &mid_lav_Cr, &mid_lav_Nb);
        fictitious(lft->x, lft->y, lft_pDel, lft_pLav, &lft_gam_Cr, &lft_gam_Nb, &lft_del_Cr, &lft_del_Nb, &lft_lav_Cr, &lft_lav_Nb);
        fictitious(rgt->x, rgt->y, rgt_pDel, rgt_pLav, &rgt_gam_Cr, &rgt_gam_Nb, &rgt_del_Cr, &rgt_del_Nb, &rgt_lav_Cr, &rgt_lav_Nb);
        fictitious(bot->x, bot->y, bot_pDel, bot_pLav, &bot_gam_Cr, &bot_gam_Nb, &bot_del_Cr, &bot_del_Nb, &bot_lav_Cr, &bot_lav_Nb);
        fictitious(top->x, top->y, top_pDel, top_pLav, &top_gam_Cr, &top_gam_Nb, &top_del_Cr, &top_del_Nb, &top_lav_Cr, &top_lav_Nb);

		// Finite Differences
		// Derivation: TKR5 pp. 301--305

		fp_t divDgradU_Cr = 0.0;
		fp_t divDgradU_Nb = 0.0;
		fp_t mid_D, lft_D, rgt_D, top_D, bot_D;

		{ // k = Cr
			// TKR5p303, Eqn. 7, term 1
			mid_D = mid_pGam * D_gam_CrCr(mid->x, mid->y);
			lft_D = lft_pGam * D_gam_CrCr(lft->x, lft->y);
			rgt_D = rgt_pGam * D_gam_CrCr(rgt->x, rgt->y);
			bot_D = bot_pGam * D_gam_CrCr(bot->x, bot->y);
			top_D = top_pGam * D_gam_CrCr(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_gam_Cr, lft_gam_Cr, rgt_gam_Cr, bot_gam_Cr, top_gam_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 2
			mid_D = mid_pGam * D_gam_CrNb(mid->x, mid->y);
			lft_D = lft_pGam * D_gam_CrNb(lft->x, lft->y);
			rgt_D = rgt_pGam * D_gam_CrNb(rgt->x, rgt->y);
			bot_D = bot_pGam * D_gam_CrNb(bot->x, bot->y);
			top_D = top_pGam * D_gam_CrNb(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_gam_Nb, lft_gam_Nb, rgt_gam_Nb, bot_gam_Nb, top_gam_Nb, dx, dy);

			// TKR5p303, Eqn. 7, term 3
			mid_D = mid_pDel * D_del_CrCr(mid->x, mid->y);
			lft_D = lft_pDel * D_del_CrCr(lft->x, lft->y);
			rgt_D = rgt_pDel * D_del_CrCr(rgt->x, rgt->y);
			bot_D = bot_pDel * D_del_CrCr(bot->x, bot->y);
			top_D = top_pDel * D_del_CrCr(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_del_Cr, lft_del_Cr, rgt_del_Cr, bot_del_Cr, top_del_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 4
			mid_D = mid_pDel * D_del_CrNb(mid->x, mid->y);
			lft_D = lft_pDel * D_del_CrNb(lft->x, lft->y);
			rgt_D = rgt_pDel * D_del_CrNb(rgt->x, rgt->y);
			bot_D = bot_pDel * D_del_CrNb(bot->x, bot->y);
			top_D = top_pDel * D_del_CrNb(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_del_Nb, lft_del_Nb, rgt_del_Nb, bot_del_Nb, top_del_Nb, dx, dy);

			// TKR5p303, Eqn. 7, term 5
			mid_D = mid_pLav * D_lav_CrCr(mid->x, mid->y);
			lft_D = lft_pLav * D_lav_CrCr(lft->x, lft->y);
			rgt_D = rgt_pLav * D_lav_CrCr(rgt->x, rgt->y);
			bot_D = bot_pLav * D_lav_CrCr(bot->x, bot->y);
			top_D = top_pLav * D_lav_CrCr(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_lav_Cr, lft_lav_Cr, rgt_lav_Cr, bot_lav_Cr, top_lav_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 6
			mid_D = mid_pLav * D_lav_CrNb(mid->x, mid->y);
			lft_D = lft_pLav * D_lav_CrNb(lft->x, lft->y);
			rgt_D = rgt_pLav * D_lav_CrNb(rgt->x, rgt->y);
			bot_D = bot_pLav * D_lav_CrNb(bot->x, bot->y);
			top_D = top_pLav * D_lav_CrNb(top->x, top->y);
			divDgradU_Cr += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_lav_Nb, lft_lav_Nb, rgt_lav_Nb, bot_lav_Nb, top_lav_Nb, dx, dy);
		}
		{ // k = Nb
			// TKR5p303, Eqn. 7, term 1
			mid_D = mid_pGam * D_gam_NbCr(mid->x, mid->y);
			lft_D = lft_pGam * D_gam_NbCr(lft->x, lft->y);
			rgt_D = rgt_pGam * D_gam_NbCr(rgt->x, rgt->y);
			bot_D = bot_pGam * D_gam_NbCr(bot->x, bot->y);
			top_D = top_pGam * D_gam_NbCr(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_gam_Cr, lft_gam_Cr, rgt_gam_Cr, bot_gam_Cr, top_gam_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 2
			mid_D = mid_pGam * D_gam_NbNb(mid->x, mid->y);
			lft_D = lft_pGam * D_gam_NbNb(lft->x, lft->y);
			rgt_D = rgt_pGam * D_gam_NbNb(rgt->x, rgt->y);
			bot_D = bot_pGam * D_gam_NbNb(bot->x, bot->y);
			top_D = top_pGam * D_gam_NbNb(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_gam_Nb, lft_gam_Nb, rgt_gam_Nb, bot_gam_Nb, top_gam_Nb, dx, dy);

			// TKR5p303, Eqn. 7, term 3
			mid_D = mid_pDel * D_del_NbCr(mid->x, mid->y);
			lft_D = lft_pDel * D_del_NbCr(lft->x, lft->y);
			rgt_D = rgt_pDel * D_del_NbCr(rgt->x, rgt->y);
			bot_D = bot_pDel * D_del_NbCr(bot->x, bot->y);
			top_D = top_pDel * D_del_NbCr(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_del_Cr, lft_del_Cr, rgt_del_Cr, bot_del_Cr, top_del_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 4
			mid_D = mid_pDel * D_del_NbNb(mid->x, mid->y);
			lft_D = lft_pDel * D_del_NbNb(lft->x, lft->y);
			rgt_D = rgt_pDel * D_del_NbNb(rgt->x, rgt->y);
			bot_D = bot_pDel * D_del_NbNb(bot->x, bot->y);
			top_D = top_pDel * D_del_NbNb(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_del_Nb, lft_del_Nb, rgt_del_Nb, bot_del_Nb, top_del_Nb, dx, dy);

			// TKR5p303, Eqn. 7, term 5
			mid_D = mid_pLav * D_lav_NbCr(mid->x, mid->y);
			lft_D = lft_pLav * D_lav_NbCr(lft->x, lft->y);
			rgt_D = rgt_pLav * D_lav_NbCr(rgt->x, rgt->y);
			bot_D = bot_pLav * D_lav_NbCr(bot->x, bot->y);
			top_D = top_pLav * D_lav_NbCr(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_lav_Cr, lft_lav_Cr, rgt_lav_Cr, bot_lav_Cr, top_lav_Cr, dx, dy);

			// TKR5p303, Eqn. 7, term 6
			mid_D = mid_pLav * D_lav_NbNb(mid->x, mid->y);
			lft_D = lft_pLav * D_lav_NbNb(lft->x, lft->y);
			rgt_D = rgt_pLav * D_lav_NbNb(rgt->x, rgt->y);
			bot_D = bot_pLav * D_lav_NbNb(bot->x, bot->y);
			top_D = top_pLav * D_lav_NbNb(top->x, top->y);
			divDgradU_Nb += discrete_laplacian(mid_D, lft_D, rgt_D, bot_D, top_D,
											   mid_lav_Nb, lft_lav_Nb, rgt_lav_Nb, bot_lav_Nb, top_lav_Nb, dx, dy);
		}

		/* record value */
		if (dst_y < ny && dst_x < nx) {
			d_conc_Cr_new[nx * dst_y + dst_x] = divDgradU_Cr;
			d_conc_Nb_new[nx * dst_y + dst_x] = divDgradU_Nb;
		}
	}
}

void device_laplacian(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t dx, const fp_t dy)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	const size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(double4);

	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	chemical_convolution_kernel <<< num_tiles, tile_size, buf_size>>> (
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->conc_Cr_old, dev->conc_Cr_new,
	    dev->conc_Nb_old, dev->conc_Nb_new,
	    nx, ny, nm,
	    dx, dy);
}

__device__ void composition_kernel(const fp_t& d_conc_Cr_old, const fp_t& d_conc_Nb_old,
                                   const fp_t& d_frac_del,    const fp_t& d_frac_lav,
                                   fp_t& d_conc_Cr_new,       fp_t& d_conc_Nb_new,
                                   const fp_t dt)
{
	/* Cahn-Hilliard equations of motion for composition */
	const fp_t divDgradU_Cr = d_conc_Cr_new;
	const fp_t divDgradU_Nb = d_conc_Nb_new;

	d_conc_Cr_new = d_conc_Cr_old + dt * divDgradU_Cr;
	d_conc_Nb_new = d_conc_Nb_old + dt * divDgradU_Nb;
}

__global__ void cahn_hilliard_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                     fp_t* d_phi_del_old, fp_t* d_phi_lav_old,
                                     fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                     const int nx, const int ny, const int nm,
                                     const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		/* Cahn-Hilliard equations of motion for composition */
		composition_kernel(d_conc_Cr_old[idx],      d_conc_Nb_old[idx],
		                   d_p(d_phi_del_old[idx]), d_p(d_phi_lav_old[idx]),
		                   d_conc_Cr_new[idx],      d_conc_Nb_new[idx],
		                   dt);
	}
}

__device__ void delta_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                             fp_t& phi_del_new,
                             const fp_t inv_fict_det,
                             const fp_t pDel,        const fp_t pLav,
                             const fp_t dgGdxCr,     const fp_t dgGdxNb,
                             const fp_t gam_Cr,      const fp_t gam_Nb,
                             const fp_t gam_nrg,     const fp_t alpha,
                             const fp_t dt)
{
	// Derivation: TKR5p281, Eqn. (14)

	const fp_t pGam = 1.0 - pDel - pLav;
	const fp_t del_Cr = d_fict_del_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, pDel, pGam, pLav);
	const fp_t del_Nb = d_fict_del_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, pDel, pGam, pLav);
	const fp_t del_nrg = d_g_del(del_Cr, del_Nb);

	/* pressure */
	const fp_t P_del = gam_nrg - del_nrg - dgGdxCr * (gam_Cr - del_Cr) - dgGdxNb * (gam_Nb - del_Nb);

	/* variational derivative */
	const fp_t dFdPhi_del = -d_pPrime(phi_del_old) * P_del
	                        + 2.0 * d_Omeg[0] * phi_del_old * (phi_del_old - 1.0) * (2.0 * phi_del_old - 1.0)
	                        + 2.0 * alpha * phi_del_old * phi_lav_old * phi_lav_old
	                        - d_Kapp[0] * phi_del_new;

	/* Allen-Cahn equation of motion for delta phase */
	phi_del_new = phi_del_old - dt * d_Lmob[0] * dFdPhi_del;
}

__device__ void laves_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                             fp_t& phi_lav_new,
                             const fp_t inv_fict_det,
                             const fp_t pDel,        const fp_t pLav,
                             const fp_t dgGdxCr,     const fp_t dgGdxNb,
                             const fp_t gam_Cr,      const fp_t gam_Nb,
                             const fp_t gam_nrg,     const fp_t alpha,
                             const fp_t dt)
{
	// Derivation: TKR5p281, Eqn. (14)

	const fp_t pGam = 1.0 - pDel - pLav;
	const fp_t lav_Cr = d_fict_lav_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, pDel, pGam, pLav);
	const fp_t lav_Nb = d_fict_lav_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, pDel, pGam, pLav);
	const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

	/* pressure */
	const fp_t P_lav = gam_nrg - lav_nrg - dgGdxCr * (gam_Cr - lav_Cr) - dgGdxNb * (gam_Nb - lav_Nb);

	/* variational derivative */
	const fp_t dFdPhi_lav = -d_pPrime(phi_lav_old) * P_lav
	                        + 2.0 * d_Omeg[1] * phi_lav_old * (phi_lav_old - 1.0) * (2.0 * phi_lav_old - 1.0)
	                        + 2.0 * alpha * phi_lav_old * phi_del_old * phi_del_old
	                        - d_Kapp[1] * phi_lav_new;

	/* Allen-Cahn equation of motion for Laves phase */
	phi_lav_new = phi_lav_old - dt * d_Lmob[1] * dFdPhi_lav;
}

__global__ void allen_cahn_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                  fp_t* d_phi_del_old, fp_t* d_phi_lav_old,
                                  fp_t* d_phi_del_new, fp_t* d_phi_lav_new,
                                  const int nx, const int ny, const int nm,
                                  const fp_t alpha,
                                  const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		const fp_t pDel = d_p(d_phi_del_old[idx]);
		const fp_t pLav = d_p(d_phi_lav_old[idx]);
		const fp_t pGam = 1.0 - pDel - pLav;
		const fp_t inv_fict_det = d_inv_fict_det(pDel, pGam, pLav);
		const fp_t gam_Cr = d_fict_gam_Cr(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], pDel, pGam, pLav);
		const fp_t gam_Nb = d_fict_gam_Nb(inv_fict_det, d_conc_Cr_old[idx], d_conc_Nb_old[idx], pDel, pGam, pLav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(gam_Cr, gam_Nb);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(gam_Cr, gam_Nb);
		const fp_t dgGdxNb = d_dg_gam_dxNb(gam_Cr, gam_Nb);

		/* Allen-Cahn equations of motion for phase */
		delta_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_del_new[idx], inv_fict_det, pDel, pLav, dgGdxCr, dgGdxNb,
		             gam_Cr, gam_Nb, gam_nrg, alpha, dt);

		laves_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_lav_new[idx], inv_fict_det, pDel, pLav, dgGdxCr, dgGdxNb,
		             gam_Cr, gam_Nb, gam_nrg, alpha, dt);
	}
}

void device_evolution(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t alpha,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	cahn_hilliard_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    nx, ny, nm,
	    dt);

	allen_cahn_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->phi_del_new, dev->phi_lav_new,
	    nx, ny, nm,
	    alpha,
	    dt);
}

__global__ void init_prng_kernel(hiprandState* d_prng, const int nx, const int ny)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;
	const int idx = nx * y + x;

	if (x < nx && y < ny)
		hiprand_init((unsigned long long)clock() + idx, x, 0, &(d_prng[idx]));
}

void device_init_prng(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	init_prng_kernel <<< num_tiles, tile_size>>> (
	    dev->prng, nx, ny);
}

__device__ void embed_OPC_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                 fp_t* d_phi_del, fp_t* d_phi_lav,
                                 const int nx, const int ny,
                                 const int x, const int y, const int idx,
                                 const fp_t xCr,
                                 const fp_t xNb,
                                 const fp_t par_xe_Cr,
                                 const fp_t par_xe_Nb,
                                 const fp_t R_precip)
{
	const fp_t R_depletion_Cr = fp_t(R_precip) * sqrt((par_xe_Cr - xCr) / (xCr - d_xe_gam_Cr()));
	const fp_t R_depletion_Nb = fp_t(R_precip) * sqrt((par_xe_Nb - xNb) / (xNb - d_xe_gam_Nb()));

	for (int i = -R_precip; i < R_precip; i++) {
		for (int j = -R_precip; j < R_precip; j++) {
			const int idn = nx * (y + j) + (x + i);
			const fp_t r = sqrt(fp_t(i * i + j * j));
			if (idn >= 0 && idn < nx * ny) {
				if (r <= R_precip) {
					d_conc_Cr[idn] = par_xe_Cr;
					d_conc_Nb[idn] = par_xe_Nb;
					d_phi_del[idn] = 1.;
				} else {
					if (r <= R_depletion_Cr)
						d_conc_Cr[idn] = d_xe_gam_Cr();
					if (r <= R_depletion_Nb)
						d_conc_Nb[idn] = d_xe_gam_Nb();
				}
			}
		}
	}
}

__global__ void nucleation_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                  fp_t* d_phi_del, fp_t* d_phi_lav,
                                  hiprandState* d_prng,
                                  const int nx, const int ny, const int nm,
                                  const fp_t sigma_del, const fp_t sigma_lav,
                                  const fp_t lattice_const,
                                  const fp_t ifce_width,
                                  const fp_t dx, const fp_t dy, const fp_t dz,
                                  const fp_t dt)
{
	/* determine indices on which to operate */
	const int x = blockDim.x * blockIdx.x + threadIdx.x;
	const int y = blockDim.y * blockIdx.y + threadIdx.y;

	const fp_t dV = dx * dy * dz;
	const fp_t Vatom = 0.25 * lattice_const * lattice_const * lattice_const; // m³/atom, assuming FCC
	const fp_t n_gam = dV / Vatom; // atoms, assuming FCC

	fp_t phi_pre = 0.;
	fp_t dG_chem = 0.;
	fp_t R_precip, R_star;
	fp_t P_nuc;
	fp_t rand_pre;

	// Scan neighborhood for existing precipitates
	if (x < nx && y < ny) {
		const fp_t R = 1.75e-9 / dx;

		for (int i = -R; i < R; i++) {
			for (int j = -R; j < R; j++) {
				const int idn = nx * (y + j) + (x + i);
				const fp_t r = sqrt(fp_t(i * i + j * j));
				if (idn >= 0 &&
				    idn < nx * ny &&
				    i * i + j * j < R * R)
					phi_pre = max(phi_pre, d_p(d_phi_del[idn]) + d_p(d_phi_lav[idn]));
			}
		}
	}
	__syncthreads();

	if (x < nx && y < ny && phi_pre < 1e-10) {
		const int idx = nx * y + x;
		const fp_t xCr = d_conc_Cr[idx];
		const fp_t xNb = d_conc_Nb[idx];
		const fp_t pDel = d_p(d_phi_del[idx]);
		const fp_t pLav = d_p(d_phi_lav[idx]);
        const fp_t pGam = 1.0 - pDel - pLav;

		// Test a delta particle
		d_nucleation_driving_force_delta(xCr, xNb, &dG_chem);
		d_nucleation_probability_sphere(xCr, xNb,
		                                dG_chem,
                                        pGam * (d_M_CrCr(xCr, xNb) * d_d2g_gam_dxCrCr() + d_M_CrNb(xCr, xNb) * d_d2g_gam_dxCrNb()),
                                        pGam * (d_M_NbCr(xCr, xNb) * d_d2g_gam_dxNbCr() + d_M_NbNb(xCr, xNb) * d_d2g_gam_dxNbNb()),
		                                sigma_del,
		                                Vatom,
		                                n_gam,
		                                dV, dt,
		                                &R_star,
		                                &P_nuc);
		if (R_star > 0.) {
			R_precip = R_star / dx;
			rand_pre = P_nuc - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

			if (rand_pre > 0)
				embed_OPC_kernel(d_conc_Cr, d_conc_Nb,
				                 d_phi_del, d_phi_lav,
				                 nx, ny,
				                 x, y, idx,
				                 xCr, xNb,
				                 d_xe_del_Cr(), d_xe_del_Nb(),
				                 R_precip);
		}

		// Test a Laves particle
		d_nucleation_driving_force_laves(xCr, xNb, &dG_chem);
		d_nucleation_probability_sphere(xCr, xNb,
		                                dG_chem,
                                        pGam * (d_M_CrCr(xCr, xNb) * d_d2g_gam_dxCrCr() + d_M_CrNb(xCr, xNb) * d_d2g_gam_dxCrNb()),
                                        pGam * (d_M_NbCr(xCr, xNb) * d_d2g_gam_dxNbCr() + d_M_NbNb(xCr, xNb) * d_d2g_gam_dxNbNb()),
		                                sigma_lav,
		                                Vatom,
		                                n_gam,
		                                dV, dt,
		                                &R_star,
		                                &P_nuc);
		if (R_star > 0.) {
			R_precip = R_star / dx;
			rand_pre = P_nuc - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

			if (rand_pre > 0)
				embed_OPC_kernel(d_conc_Cr, d_conc_Nb,
				                 d_phi_lav, d_phi_lav,
				                 nx, ny,
				                 x, y, idx,
				                 xCr, xNb,
				                 d_xe_lav_Cr(), d_xe_lav_Nb(),
				                 R_precip);
		}
	}
}

void device_nucleation(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by,
                       const fp_t sigma_del, const fp_t sigma_lav,
                       const fp_t lattice_const, const fp_t ifce_width,
                       const fp_t dx, const fp_t dy, const fp_t dz,
                       const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	nucleation_kernel <<< num_tiles, tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->prng,
	    nx, ny, nm,
	    sigma_del, sigma_lav,
	    lattice_const, ifce_width,
	    dx, dy, dz, dt);
}

__global__ void dataviz_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb, fp_t* d_conc_Ni,
                               fp_t* d_phi_del, fp_t* d_phi_lav, fp_t* d_phi,
                               const int nx, const int ny)
{
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		d_conc_Ni[idx] = 1.0 - d_conc_Cr[idx] - d_conc_Nb[idx];
		d_phi[idx] = d_p(d_phi_del[idx]) + d_p(d_phi_lav[idx]);
	}
}

void device_dataviz(struct CudaData* dev, struct HostData* host,
                    const int nx, const int ny, const int nm,
                    const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	dataviz_kernel <<< num_tiles, tile_size>>>(
	    dev->conc_Cr_old, dev->conc_Nb_old, dev->conc_Ni,
	    dev->phi_del_old, dev->phi_lav_old, dev->phi,
	    nx, ny);

	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Ni[0], dev->conc_Ni, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);

	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipMemcpy(host->phi[0], dev->phi, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
}
