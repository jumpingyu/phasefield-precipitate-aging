#include "hip/hip_runtime.h"
/**********************************************************************************
 HiPerC: High Performance Computing Strategies for Boundary Value Problems
 written by Trevor Keller and available from https://github.com/usnistgov/hiperc

 This software was developed at the National Institute of Standards and Technology
 by employees of the Federal Government in the course of their official duties.
 Pursuant to title 17 section 105 of the United States Code this software is not
 subject to copyright protection and is in the public domain. NIST assumes no
 responsibility whatsoever for the use of this software by other parties, and makes
 no guarantees, expressed or implied, about its quality, reliability, or any other
 characteristic. We would appreciate acknowledgement if the software is used.

 This software can be redistributed and/or modified freely provided that any
 derivative works bear some notice that they are derived from it, and any modified
 versions bear some notice that they have been modified.

 Questions/comments to Trevor Keller (trevor.keller@nist.gov)
 **********************************************************************************/

/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#include "cuda_data.h"
#include "numerics.h"
#include "mesh.h"

#include "cuda_kernels.cuh"
#include "parabola625.cuh"

__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new,
                                   const int nx, const int ny, const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm/2;
	const int src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)], NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_conc_new[nx * dst_y + dst_x] = value;
		}
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

__device__ void composition_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                                         fp_t& conc_Cr_new,       fp_t& conc_Nb_new,
                                   const fp_t& D_CrCr,      const fp_t& D_CrNb,
                                   const fp_t& D_NbCr,      const fp_t& D_NbNb,
                                   const fp_t& dt)
{
	/* Cahn-Hilliard equations of motion for composition */
	const fp_t lap_mu_Cr = D_CrCr * conc_Cr_new
	                     + D_NbCr * conc_Nb_new;
	const fp_t lap_mu_Nb = D_CrNb * conc_Cr_new
	                     + D_NbNb * conc_Nb_new;

	conc_Cr_new = conc_Cr_old + dt * lap_mu_Cr;
	conc_Nb_new = conc_Nb_old + dt * lap_mu_Nb;
}

__device__ void delta_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
	                         const fp_t& phi_del_old, const fp_t& phi_lav_old,
	                               fp_t& phi_del_new, const fp_t& inv_fict_det,
	                         const fp_t& f_del,       const fp_t& f_lav,
	                         const fp_t& dgGdxCr,     const fp_t& dgGdxNb,
	                         const fp_t& gam_Cr,      const fp_t& gam_Nb,
	                         const fp_t& gam_nrg,     const fp_t& alpha,
                             const fp_t& kappa,       const fp_t& omega,
                             const fp_t& M_del,       const fp_t& dt)
{
	const fp_t del_Cr = d_fict_del_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, 1.-f_del-f_lav, f_lav);
	const fp_t del_Nb = d_fict_del_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, 1.-f_del-f_lav, f_lav);
	const fp_t del_nrg = d_g_del(del_Cr, del_Nb);

    /* pressure */
	const fp_t P_del = gam_nrg - del_nrg - dgGdxCr * (gam_Cr - del_Cr) - dgGdxNb * (gam_Nb - del_Nb);

	/* variational derivative */
	const fp_t dFdPhi_del = -d_hprime(phi_del_old) * P_del
	                      + 2. * omega * phi_del_old * (phi_del_old - 1.) * (2. * phi_del_old - 1.)
	                      + 2. * alpha * phi_del_old * (phi_lav_old * phi_lav_old)
	                      - kappa * phi_del_new;

	/* Allen-Cahn equation of motion for delta phase */
	phi_del_new = phi_del_old - dt * M_del * dFdPhi_del;
}

__device__ void laves_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
	                         const fp_t& phi_del_old, const fp_t& phi_lav_old,
	                               fp_t& phi_lav_new, const fp_t& inv_fict_det,
	                         const fp_t& f_del,       const fp_t& f_lav,
	                         const fp_t& dgGdxCr,     const fp_t& dgGdxNb,
	                         const fp_t& gam_Cr,      const fp_t& gam_Nb,
	                         const fp_t& gam_nrg,     const fp_t& alpha,
                             const fp_t& kappa,       const fp_t& omega,
                             const fp_t& M_lav,       const fp_t& dt)
{
	const fp_t lav_Cr = d_fict_lav_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_lav, 1.-f_del-f_lav, f_lav);
	const fp_t lav_Nb = d_fict_lav_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_lav, 1.-f_del-f_lav, f_lav);
	const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

    /* pressure */
	const fp_t P_lav = gam_nrg - lav_nrg - dgGdxCr * (gam_Cr - lav_Cr) - dgGdxNb * (gam_Nb - lav_Nb);

	/* variational derivative */
	const fp_t dFdPhi_lav = -d_hprime(phi_lav_old) * P_lav
	                      + 2. * omega * phi_lav_old * (phi_lav_old - 1.) * (2. * phi_lav_old - 1.)
	                      + 2. * alpha * phi_lav_old * (phi_del_old * phi_del_old)
	                      - kappa * phi_lav_new;

	/* Allen-Cahn equation of motion for delta phase */
	phi_lav_new = phi_lav_old - dt * M_lav * dFdPhi_lav;
}

__device__ void fictitious_kernel(const fp_t& phi_del_new, const fp_t& phi_lav_new,
                                  const fp_t& conc_Cr_new, const fp_t& conc_Nb_new,
                                        fp_t& gam_Cr_new,        fp_t& gam_Nb_new)
{
	const fp_t f_del = d_h(phi_del_new);
	const fp_t f_lav = d_h(phi_lav_new);
	const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);
	gam_Cr_new = d_fict_gam_Cr(inv_fict_det, conc_Cr_new, conc_Nb_new,
	                                  f_del, 1.-f_del-f_lav, f_lav);
	gam_Nb_new = d_fict_gam_Nb(inv_fict_det, conc_Cr_new, conc_Nb_new,
	                                  f_del, 1.-f_del-f_lav, f_lav);
}
__global__ void evolution_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                 fp_t* d_phi_del_old,
                                 fp_t* d_phi_lav_old,
                                 fp_t* d_gam_Cr_old,  fp_t* d_gam_Nb_old,
                                 fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                 fp_t* d_phi_del_new,
                                 fp_t* d_phi_lav_new,
                                 fp_t* d_gam_Cr_new,  fp_t* d_gam_Nb_new,
                                 const int nx, const int ny, const int nm,
                                 const fp_t D_CrCr, const fp_t D_CrNb,
                                 const fp_t D_NbCr, const fp_t D_NbNb,
                                 const fp_t alpha, const fp_t kappa, const fp_t omega,
                                 const fp_t M_del, const fp_t M_lav,
                                 const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr_old[idx], d_gam_Nb_old[idx]);

		/* Cahn-Hilliard equations of motion for composition */
        composition_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx],
                           d_conc_Cr_new[idx], d_conc_Nb_new[idx],
                           D_CrCr, D_CrNb, D_NbCr, D_NbNb, dt);

		/* Allen-Cahn equations of motion for phase */
		delta_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
                     d_phi_del_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
                     d_gam_Cr_old[idx], d_gam_Nb_old[idx], gam_nrg, alpha, kappa, omega,
                     M_del, dt);
		laves_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
                     d_phi_lav_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
                     d_gam_Cr_old[idx], d_gam_Nb_old[idx], gam_nrg, alpha, kappa, omega,
                     M_lav, dt);
    }

	/* wait for all threads to finish writing */
	__syncthreads();

    /* fictitious compositions */
	if (x < nx && y < ny) {
		fictitious_kernel(d_phi_del_new[idx], d_phi_lav_new[idx],
                          d_conc_Cr_new[idx], d_conc_Nb_new[idx],
                          d_gam_Cr_new[idx],  d_gam_Nb_new[idx]);
	}

	/* wait for all threads to finish writing */
	__syncthreads();
}

void device_boundaries(struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old,
	    dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    nx, ny, nm
	);
}

void device_laplacian_boundaries(struct CudaData* dev,
                            const int nx, const int ny, const int nm,
                            const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);

	boundary_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new,
	    dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm
	);
}

void device_laplacian(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Cr_old, dev->conc_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->conc_Nb_old, dev->conc_Nb_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Cr_old, dev->gam_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles,tile_size,buf_size>>> (
	    dev->gam_Nb_old, dev->gam_Nb_new, nx, ny, nm);
}

void device_evolution(struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t D_CrCr, const fp_t D_CrNb,
                      const fp_t D_NbCr, const fp_t D_NbNb,
                      const fp_t alpha, const fp_t kappa, const fp_t omega,
                      const fp_t M_del, const fp_t M_lav,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(ceil(float(nx) / (tile_size.x - nm + 1)),
	               ceil(float(ny) / (tile_size.y - nm + 1)),
	               1);
	evolution_kernel<<<num_tiles,tile_size>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr_old, dev->gam_Nb_old,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr_new, dev->gam_Nb_new,
	    nx, ny, nm,
	    D_CrCr, D_CrNb,
	    D_NbCr, D_NbNb,
	    alpha, kappa, omega,
	    M_del, M_lav,
	    dt);
}

void read_out_result(struct CudaData* dev, struct HostData* host, const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Cr_new[0], dev->gam_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Nb_new[0], dev->gam_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}
