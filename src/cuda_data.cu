/**
 \file  cuda_data.cu
 \brief Implementation of functions to create and destroy CudaData struct
*/

#include <hiprand.h>
#include "cuda_data.h"
#include "cuda_kernels.cuh"

void init_cuda(struct HostData* host,
               const int nx, const int ny, const int nm, struct CudaData* dev)
{
	/* allocate memory on device */
	hipMalloc((void**) &(dev->conc_Cr_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Cr_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_new), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->phi_del_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_del_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_new), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->gam_Cr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->gam_Nb), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->lap_gam_Cr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->lap_gam_Nb), nx * ny * sizeof(fp_t));

    hipMalloc((void**) &(dev->prng), nx * ny * sizeof(hiprandState));

	/* transfer mask and boundary conditions to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask), host->mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer data from host in to GPU */
	hipMemcpy(dev->conc_Cr_old, host->conc_Cr_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->conc_Nb_old, host->conc_Nb_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);

	hipMemcpy(dev->phi_del_old, host->phi_del_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->phi_lav_old, host->phi_lav_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);

	hipMemcpy(dev->gam_Cr, host->gam_Cr[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->gam_Nb, host->gam_Nb[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
}

void free_cuda(struct CudaData* dev)
{
	/* free memory on device */
	hipFree(dev->conc_Cr_old);
	hipFree(dev->conc_Cr_new);
	hipFree(dev->conc_Nb_old);
	hipFree(dev->conc_Nb_new);

	hipFree(dev->phi_del_old);
	hipFree(dev->phi_del_new);
	hipFree(dev->phi_lav_old);
	hipFree(dev->phi_lav_new);

	hipFree(dev->gam_Cr);
	hipFree(dev->gam_Nb);
	hipFree(dev->lap_gam_Cr);
	hipFree(dev->lap_gam_Nb);

    hipFree(dev->prng);
}
