/**
 \file  data.cu
 \brief Implementation of functions to create and destroy CudaData struct
*/

#include <hiprand.h>
#include "data.cuh"
#include "discretization.cuh"

void init_cuda(struct HostData* host,
               const int nx, const int ny, const int nm,
			   const fp_t* kappa, const fp_t* omega, const fp_t* Lmob, 
			   struct CudaData* dev)
{
	/* allocate memory on device */
	hipMalloc((void**) &(dev->prng), nx * ny * sizeof(hiprandState));

	hipStreamCreate(&(dev->str_A));
	hipStreamCreate(&(dev->str_B));
	hipStreamCreate(&(dev->str_C));
	hipStreamCreate(&(dev->str_D));

	hipEventCreate(&(dev->ev_A));
	hipEventCreate(&(dev->ev_B));
	hipEventCreate(&(dev->ev_C));
	hipEventCreate(&(dev->ev_D));

	hipMalloc((void**) &(dev->conc_Cr_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Cr_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_new), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->phi_del_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_del_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_old), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi_lav_new), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->phi),         nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->conc_Cr_gam), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Cr_del), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Cr_lav), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_gam), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_del), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->conc_Nb_lav), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->conc_Ni), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->mob_gam_CrCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_gam_CrNb), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_gam_NbCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_gam_NbNb), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->mob_del_CrCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_del_CrNb), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_del_NbCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_del_NbNb), nx * ny * sizeof(fp_t));

	hipMalloc((void**) &(dev->mob_lav_CrCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_lav_CrNb), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_lav_NbCr), nx * ny * sizeof(fp_t));
	hipMalloc((void**) &(dev->mob_lav_NbNb), nx * ny * sizeof(fp_t));

	/* transfer mask to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_mask), host->mask_lap[0], nm * nm * sizeof(fp_t));

	/* transfer mobility data to protected memory on GPU */
	hipMemcpyToSymbol(HIP_SYMBOL(d_Kapp), kappa, NP * sizeof(fp_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_Omeg), omega, NP * sizeof(fp_t));
	hipMemcpyToSymbol(HIP_SYMBOL(d_Lmob), Lmob,  NP * sizeof(fp_t));

	/* transfer data from host in to GPU */
	hipMemcpy(dev->conc_Cr_old, host->conc_Cr_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->conc_Nb_old, host->conc_Nb_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);

	hipMemcpy(dev->phi_del_old, host->phi_del_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
	hipMemcpy(dev->phi_lav_old, host->phi_lav_old[0], nx * ny * sizeof(fp_t),
	           hipMemcpyHostToDevice);
}

void free_cuda(struct CudaData* dev)
{
	/* free memory on device */
	hipFree(dev->prng);

	hipStreamDestroy(dev->str_A);
	hipStreamDestroy(dev->str_B);
	hipStreamDestroy(dev->str_C);
	hipStreamDestroy(dev->str_D);

	hipEventDestroy(dev->ev_A);
	hipEventDestroy(dev->ev_B);
	hipEventDestroy(dev->ev_C);
	hipEventDestroy(dev->ev_D);

	hipFree(dev->conc_Cr_old);
	hipFree(dev->conc_Cr_new);
	hipFree(dev->conc_Nb_old);
	hipFree(dev->conc_Nb_new);

	hipFree(dev->phi_del_old);
	hipFree(dev->phi_del_new);
	hipFree(dev->phi_lav_old);
	hipFree(dev->phi_lav_new);
	hipFree(dev->phi);

	hipFree(dev->conc_Cr_gam);
	hipFree(dev->conc_Cr_del);
	hipFree(dev->conc_Cr_lav);
	hipFree(dev->conc_Nb_gam);
	hipFree(dev->conc_Nb_del);
	hipFree(dev->conc_Nb_lav);

	hipFree(dev->conc_Ni);

	hipFree(dev->mob_gam_CrCr);
	hipFree(dev->mob_gam_CrNb);
	hipFree(dev->mob_gam_NbCr);
	hipFree(dev->mob_gam_NbNb);

	hipFree(dev->mob_del_CrCr);
	hipFree(dev->mob_del_CrNb);
	hipFree(dev->mob_del_NbCr);
	hipFree(dev->mob_del_NbNb);

	hipFree(dev->mob_lav_CrCr);
	hipFree(dev->mob_lav_CrNb);
	hipFree(dev->mob_lav_NbCr);
	hipFree(dev->mob_lav_NbNb);
}

void read_out_result(struct CudaData* dev, struct HostData* host,
                     const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
}
