#include "hip/hip_runtime.h"
/**
 \file  cuda_discretization.cu
 \brief Implementation of boundary condition functions with CUDA acceleration
*/

#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>

#include "cuda_data.h"
#include "numerics.h"
#include "mesh.h"

#include "cuda_kernels.cuh"
#include "parabola625.cuh"
#include "nucleation.cuh"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}


__constant__ fp_t d_mask[MAX_MASK_W * MAX_MASK_H];

float nTiles(int domain_size, int tile_loc, int mask_size)
{
	return ceil(float(domain_size) / float(tile_loc - mask_size + 1));
}

__global__ void boundary_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                fp_t* d_phi_del, fp_t* d_phi_lav,
                                fp_t* d_gam_Cr,  fp_t* d_gam_Nb,
                                const int nx,
                                const int ny,
                                const int nm)
{
	/* determine indices on which to operate */
	const int tx = threadIdx.x;
	const int ty = threadIdx.y;

	const int row = blockDim.y * blockIdx.y + ty;
	const int col = blockDim.x * blockIdx.x + tx;

	/* apply no-flux boundary conditions: inside to out, sequence matters */

	for (int offset = 0; offset < nm/2; offset++) {
		const int ilo = nm/2 - offset;
		const int ihi = nx - 1 - nm/2 + offset;
		const int jlo = nm/2 - offset;
		const int jhi = ny - 1 - nm/2 + offset;

		if (ilo-1 == col && row < ny) {
			/* left condition */
			d_conc_Cr[row * nx + ilo-1] = d_conc_Cr[row * nx + ilo];
			d_conc_Nb[row * nx + ilo-1] = d_conc_Nb[row * nx + ilo];
			d_phi_del[row * nx + ilo-1] = d_phi_del[row * nx + ilo];
			d_phi_lav[row * nx + ilo-1] = d_phi_lav[row * nx + ilo];
			d_gam_Cr[ row * nx + ilo-1] = d_gam_Cr[ row * nx + ilo];
			d_gam_Nb[ row * nx + ilo-1] = d_gam_Nb[ row * nx + ilo];
		}
		if (ihi+1 == col && row < ny) {
			/* right condition */
			d_conc_Cr[row * nx + ihi+1] = d_conc_Cr[row * nx + ihi];
			d_conc_Nb[row * nx + ihi+1] = d_conc_Nb[row * nx + ihi];
			d_phi_del[row * nx + ihi+1] = d_phi_del[row * nx + ihi];
			d_phi_lav[row * nx + ihi+1] = d_phi_lav[row * nx + ihi];
			d_gam_Cr[ row * nx + ihi+1] = d_gam_Cr[ row * nx + ihi];
			d_gam_Nb[ row * nx + ihi+1] = d_gam_Nb[ row * nx + ihi];
		}
		if (jlo-1 == row && col < nx) {
			/* bottom condition */
			d_conc_Cr[(jlo-1) * nx + col] = d_conc_Cr[jlo * nx + col];
			d_conc_Nb[(jlo-1) * nx + col] = d_conc_Nb[jlo * nx + col];
			d_phi_del[(jlo-1) * nx + col] = d_phi_del[jlo * nx + col];
			d_phi_lav[(jlo-1) * nx + col] = d_phi_lav[jlo * nx + col];
			d_gam_Cr[ (jlo-1) * nx + col] = d_gam_Cr[ jlo * nx + col];
			d_gam_Nb[ (jlo-1) * nx + col] = d_gam_Nb[ jlo * nx + col];
		}
		if (jhi+1 == row && col < nx) {
			/* top condition */
			d_conc_Cr[(jhi+1) * nx + col] = d_conc_Cr[jhi * nx + col];
			d_conc_Nb[(jhi+1) * nx + col] = d_conc_Nb[jhi * nx + col];
			d_gam_Cr[ (jhi+1) * nx + col] = d_gam_Cr[ jhi * nx + col];
			d_gam_Nb[ (jhi+1) * nx + col] = d_gam_Nb[ jhi * nx + col];
			d_phi_del[(jhi+1) * nx + col] = d_phi_del[jhi * nx + col];
			d_phi_lav[(jhi+1) * nx + col] = d_phi_lav[jhi * nx + col];
		}

		__syncthreads();
	}
}

void device_boundaries(hipStream_t& stream,
                       struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel<<<num_tiles, tile_size, 0, stream>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny, nm
	);
}

void device_laplacian_boundaries(hipStream_t& stream,
                                 struct CudaData* dev,
                                 const int nx, const int ny, const int nm,
                                 const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	boundary_kernel<<<num_tiles, tile_size, 0, stream>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->lap_gam_Cr,  dev->lap_gam_Nb,
	    nx, ny, nm
	);
}

__global__ void convolution_kernel(fp_t* d_conc_old, fp_t* d_conc_new,
                                   const int nx, const int ny, const int nm)
{
	/* source and tile width include the halo cells */
	const int src_nx = blockDim.x;
	const int src_ny = blockDim.y;
	const int til_nx = src_nx;

	/* destination width excludes the halo cells */
	const int dst_nx = src_nx - nm + 1;
	const int dst_ny = src_ny - nm + 1;

	/* determine tile indices on which to operate */
	const int til_x = threadIdx.x;
	const int til_y = threadIdx.y;

	const int dst_x = blockIdx.x * dst_nx + til_x;
	const int dst_y = blockIdx.y * dst_ny + til_y;

	const int src_x = dst_x - nm/2;
	const int src_y = dst_y - nm/2;

	/* copy tile: __shared__ gives access to all threads working on this tile */
	extern __shared__ fp_t d_conc_tile[];

	if (src_x >= 0 && src_x < nx &&
	    src_y >= 0 && src_y < ny ) {
		/* if src_y==0, then dst_y==nm/2: this is a halo row */
		d_conc_tile[til_nx * til_y + til_x] = d_conc_old[nx * src_y + src_x];
	}

	/* tile data is shared: wait for all threads to finish copying */
	__syncthreads();

	/* compute the convolution */
	if (til_x < dst_nx && til_y < dst_ny) {
		fp_t value = 0.;
		for (int j = 0; j < nm; j++) {
			for (int i = 0; i < nm; i++) {
				value += d_mask[j * nm + i] * d_conc_tile[til_nx * (til_y+j) + til_x+i];
			}
		}
		/* record value */
		/* Note: tile is centered on [til_nx*(til_y+nm/2) + (til_x+nm/2)],
		         NOT [til_nx*til_y + til_x] */
		if (dst_y < ny && dst_x < nx) {
			d_conc_new[nx * dst_y + dst_x] = value;
		}
	}
}

void device_laplacian(hipStream_t& stream,
                      struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	size_t buf_size = (tile_size.x + nm) * (tile_size.y + nm) * sizeof(fp_t);

	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->conc_Cr_old, dev->conc_Cr_new, nx, ny, nm);
	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->conc_Nb_old, dev->conc_Nb_new, nx, ny, nm);

	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->phi_del_old, dev->phi_del_new, nx, ny, nm);
	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->phi_lav_old, dev->phi_lav_new, nx, ny, nm);

	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->gam_Cr, dev->lap_gam_Cr, nx, ny, nm);
	convolution_kernel<<<num_tiles, tile_size, buf_size, stream>>> (
	    dev->gam_Nb, dev->lap_gam_Nb, nx, ny, nm);
}

__device__ void composition_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                                   const fp_t& lap_gam_Cr,  const fp_t& lap_gam_Nb,
                                         fp_t& conc_Cr_new,       fp_t& conc_Nb_new,
                                   const fp_t& D_CrCr,      const fp_t& D_CrNb,
                                   const fp_t& D_NbCr,      const fp_t& D_NbNb,
                                   const fp_t& dt)
{
	/* Cahn-Hilliard equations of motion for composition */
	const fp_t lap_mu_Cr = D_CrCr * lap_gam_Cr
	                     + D_NbCr * lap_gam_Nb;
	const fp_t lap_mu_Nb = D_CrNb * lap_gam_Cr
	                     + D_NbNb * lap_gam_Nb;

	conc_Cr_new = conc_Cr_old + dt * lap_mu_Cr;
	conc_Nb_new = conc_Nb_old + dt * lap_mu_Nb;
}

__device__ void delta_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                                   fp_t& phi_del_new, const fp_t& inv_fict_det,
                             const fp_t& f_del,       const fp_t& f_lav,
                             const fp_t& dgGdxCr,     const fp_t& dgGdxNb,
                             const fp_t& gam_Cr,      const fp_t& gam_Nb,
                             const fp_t& gam_nrg,     const fp_t& alpha,
                             const fp_t& kappa,       const fp_t& omega,
                             const fp_t& M_del,       const fp_t& dt)
{
	const fp_t f_gam = 1. - f_del - f_lav;
	const fp_t del_Cr = d_fict_del_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t del_Nb = d_fict_del_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t del_nrg = d_g_del(del_Cr, del_Nb);

	/* pressure */
	const fp_t P_del = gam_nrg - del_nrg - dgGdxCr * (gam_Cr - del_Cr) - dgGdxNb * (gam_Nb - del_Nb);

	/* variational derivative */
	const fp_t dFdPhi_del = -d_hprime(phi_del_old) * P_del
	                      + 2. * omega * phi_del_old * (phi_del_old - 1.) * (2. * phi_del_old - 1.)
	                      + 2. * alpha * phi_del_old * (phi_lav_old * phi_lav_old)
	                      - kappa * phi_del_new;

	/* Allen-Cahn equation of motion for delta phase */
	phi_del_new = phi_del_old - dt * M_del * dFdPhi_del;
}

__device__ void laves_kernel(const fp_t& conc_Cr_old, const fp_t& conc_Nb_old,
                             const fp_t& phi_del_old, const fp_t& phi_lav_old,
                                   fp_t& phi_lav_new, const fp_t& inv_fict_det,
                             const fp_t& f_del,       const fp_t& f_lav,
                             const fp_t& dgGdxCr,     const fp_t& dgGdxNb,
                             const fp_t& gam_Cr,      const fp_t& gam_Nb,
                             const fp_t& gam_nrg,     const fp_t& alpha,
                             const fp_t& kappa,       const fp_t& omega,
                             const fp_t& M_lav,       const fp_t& dt)
{
	const fp_t f_gam = 1. - f_del - f_lav;
	const fp_t lav_Cr = d_fict_lav_Cr(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t lav_Nb = d_fict_lav_Nb(inv_fict_det, conc_Cr_old, conc_Nb_old, f_del, f_gam, f_lav);
	const fp_t lav_nrg = d_g_lav(lav_Cr, lav_Nb);

	/* pressure */
	const fp_t P_lav = gam_nrg - lav_nrg - dgGdxCr * (gam_Cr - lav_Cr) - dgGdxNb * (gam_Nb - lav_Nb);

	/* variational derivative */
	const fp_t dFdPhi_lav = -d_hprime(phi_lav_old) * P_lav
	                      + 2. * omega * phi_lav_old * (phi_lav_old - 1.) * (2. * phi_lav_old - 1.)
	                      + 2. * alpha * phi_lav_old * (phi_del_old * phi_del_old)
	                      - kappa * phi_lav_new;

	/* Allen-Cahn equation of motion for Laves phase */
	phi_lav_new = phi_lav_old - dt * M_lav * dFdPhi_lav;
}

__global__ void fictitious_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                  fp_t* d_phi_del, fp_t* d_phi_lav,
                                  fp_t* d_gam_Cr,  fp_t* d_gam_Nb,
                                  const int nx, const int ny)
{
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del[idx]);
		const fp_t f_lav = d_h(d_phi_lav[idx]);
		const fp_t f_gam = 1. - f_del - f_lav;
		const fp_t inv_fict_det = d_inv_fict_det(f_del, f_gam, f_lav);

		d_gam_Cr[idx] = d_fict_gam_Cr(inv_fict_det, d_conc_Cr[idx], d_conc_Nb[idx],
		                              f_del, f_gam, f_lav);
		d_gam_Nb[idx] = d_fict_gam_Nb(inv_fict_det, d_conc_Cr[idx], d_conc_Nb[idx],
		                              f_del, f_gam, f_lav);
	}
}

void device_fictitious(hipStream_t& stream,
                       struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	fictitious_kernel<<<num_tiles, tile_size, 0, stream>>>(
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny);
}

__global__ void evolution_kernel(fp_t* d_conc_Cr_old, fp_t* d_conc_Nb_old,
                                 fp_t* d_phi_del_old, fp_t* d_phi_lav_old,
                                 fp_t* d_lap_gam_Cr,  fp_t* d_lap_gam_Nb,
                                 fp_t* d_conc_Cr_new, fp_t* d_conc_Nb_new,
                                 fp_t* d_phi_del_new, fp_t* d_phi_lav_new,
                                 fp_t* d_gam_Cr,      fp_t* d_gam_Nb,
                                 const int nx, const int ny, const int nm,
                                 const fp_t D_CrCr, const fp_t D_CrNb,
                                 const fp_t D_NbCr, const fp_t D_NbNb,
                                 const fp_t alpha, const fp_t kappa, const fp_t omega,
                                 const fp_t M_del, const fp_t M_lav,
                                 const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	/* explicit Euler solution to the equation of motion */
	if (x < nx && y < ny) {
		const fp_t f_del = d_h(d_phi_del_old[idx]);
		const fp_t f_lav = d_h(d_phi_lav_old[idx]);
		const fp_t inv_fict_det = d_inv_fict_det(f_del, 1.-f_del-f_lav, f_lav);

		/* pure phase energy */
		const fp_t gam_nrg = d_g_gam(d_gam_Cr[idx], d_gam_Nb[idx]);

		/* effective chemical potential */
		const fp_t dgGdxCr = d_dg_gam_dxCr(d_gam_Cr[idx], d_gam_Nb[idx]);
		const fp_t dgGdxNb = d_dg_gam_dxNb(d_gam_Cr[idx], d_gam_Nb[idx]);

		/* Cahn-Hilliard equations of motion for composition */
		composition_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx],
		                   d_lap_gam_Cr[idx],  d_lap_gam_Nb[idx],
		                   d_conc_Cr_new[idx], d_conc_Nb_new[idx],
		                   D_CrCr, D_CrNb, D_NbCr, D_NbNb, dt);

		/* Allen-Cahn equations of motion for phase */
		delta_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_del_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
		             d_gam_Cr[idx], d_gam_Nb[idx], gam_nrg, alpha, kappa, omega,
		             M_del, dt);
		laves_kernel(d_conc_Cr_old[idx], d_conc_Nb_old[idx], d_phi_del_old[idx], d_phi_lav_old[idx],
		             d_phi_lav_new[idx], inv_fict_det, f_del, f_lav, dgGdxCr, dgGdxNb,
		             d_gam_Cr[idx], d_gam_Nb[idx], gam_nrg, alpha, kappa, omega,
		             M_lav, dt);
	}
}

void device_evolution(hipStream_t& stream,
                      struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by,
                      const fp_t* D_Cr, const fp_t* D_Nb,
                      const fp_t alpha, const fp_t kappa, const fp_t omega,
                      const fp_t M_del, const fp_t M_lav,
                      const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	evolution_kernel<<<num_tiles, tile_size, 0, stream>>> (
	    dev->conc_Cr_old, dev->conc_Nb_old,
	    dev->phi_del_old, dev->phi_lav_old,
	    dev->lap_gam_Cr,  dev->lap_gam_Nb,
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
	    dev->gam_Cr,      dev->gam_Nb,
	    nx, ny, nm,
	    D_Cr[0], D_Cr[1],
	    D_Nb[0], D_Nb[1],
	    alpha, kappa, omega,
	    M_del, M_lav,
	    dt);
}

__global__ void init_prng_kernel(hiprandState* d_prng, const int nx, const int ny)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny)
        hiprand_init((unsigned long long)clock() + idx, x, 0, &(d_prng[idx]));
}

void device_init_prng(hipStream_t& stream,
                      struct CudaData* dev,
                      const int nx, const int ny, const int nm,
                      const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	init_prng_kernel<<<num_tiles, tile_size, 0, stream>>> (
	    dev->prng, nx, ny);
}

__global__ void nucleation_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb,
                                  fp_t* d_phi_del, fp_t* d_phi_lav,
                                  hiprandState* d_prng,
                                  const int nx, const int ny, const int nm,
                                  const fp_t D_CrCr, const fp_t D_NbNb,
                                  const fp_t sigma_del, const fp_t sigma_lav,
                                  const fp_t lattice_const, const fp_t ifce_width,
                                  const fp_t dx, const fp_t dy, const fp_t dz,
                                  const fp_t dt)
{
	/* determine indices on which to operate */
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;

    const fp_t dV = dx * dy * dz;
    const fp_t Vatom = 0.25 * lattice_const * lattice_const * lattice_const; // assuming FCC
    const fp_t n_gam = M_PI / (3. * sqrt(2.) * Vatom); // assuming FCC
    const fp_t w = ifce_width / dx;

    fp_t phi_pre = 0.;
    fp_t dG_chem = 0., pre_xCr = 0., pre_xNb = 0.;
    int R_pre;
    fp_t r_pre, r_pre_star;
    fp_t P_nuc_pre;
    fp_t rand_pre;

    if (x < nx && y < ny) {
        const fp_t rad = anticap * 1.75e-9 / dx;
        const int R = 1.25 * ceil(rad + w);

        for (int i = -R; i < R; i++) {
            for (int j = -R; j < R; j++) {
                const int idn = nx * (y + j) + (x + i);
                const fp_t r = sqrt(fp_t(i*i + j*j));
                if (idn >= 0 &&
                    idn < nx * ny &&
                    i*i + j*j < R*R)
                    phi_pre = max(phi_pre, d_h(d_phi_del[idn])
                                         + d_h(d_phi_lav[idn]));
            }
        }
    }
    __syncthreads();

    if (x < nx && y < ny && phi_pre < 1e-10) {
        const int idx = nx * y + x;
        const fp_t xCr = d_conc_Cr[idx];
        const fp_t xNb = d_conc_Nb[idx];

        // Test a delta particle
        d_nucleation_driving_force_delta(xCr, xNb, &dG_chem);
        d_nucleation_probability_sphere(xCr, xNb,
                                        pre_xCr, pre_xNb,
                                        dG_chem,
                                        D_CrCr, D_NbNb,
                                        sigma_del,
                                        Vatom,
                                        n_gam,
                                        dV, dt,
                                        &r_pre_star, &P_nuc_pre);
        if (r_pre_star > 0.) {
            r_pre = anticap * r_pre_star / dx;
            R_pre = 1.25 * ceil(r_pre + w);
            rand_pre = P_nuc_pre - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

            if (rand_pre > 0) {
                for (int i = -R_pre; i < R_pre; i++) {
                    for (int j = -R_pre; j < R_pre; j++) {
                        const int idn = nx * (y + j) + (x + i);
                        const fp_t r = sqrt(fp_t(i*i + j*j));
                        const fp_t z = r - (r_pre + w);
                        if (idn >= 0 && idn < nx * ny)
                            d_phi_del[idn] = d_interface_profile(4 * z / w);
                    }
                }
            }
        }

        // Test a Laves particle
        d_nucleation_driving_force_laves(xCr, xNb, &dG_chem);
        d_nucleation_probability_sphere(xCr, xNb,
                                        pre_xCr, pre_xNb,
                                        dG_chem,
                                        D_CrCr, D_NbNb,
                                        sigma_lav,
                                        Vatom,
                                        n_gam,
                                        dV, dt,
                                        &r_pre_star, &P_nuc_pre);
        if (r_pre_star > 0.) {
            r_pre = anticap * r_pre_star / dx;
            R_pre = 1.25 * ceil(r_pre + w);
            rand_pre = P_nuc_pre - (fp_t)hiprand_uniform_double(&(d_prng[idx]));

            if (rand_pre > 0) {
                for (int i = -R_pre; i < R_pre; i++) {
                    for (int j = -R_pre; j < R_pre; j++) {
                        const int idn = nx * (y + j) + (x + i);
                        const fp_t r = sqrt(fp_t(i*i + j*j));
                        const fp_t z = r - (r_pre + w);
                        if (idn >= 0 && idn < nx * ny)
                            d_phi_lav[idn] = d_interface_profile(4 * z / w);
                    }
                }
            }
        }
    }
}

void device_nucleation(hipStream_t& stream,
                       struct CudaData* dev,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by,
                       const fp_t* D_Cr, const fp_t* D_Nb,
                       const fp_t sigma_del, const fp_t sigma_lav,
                       const fp_t lattice_const, const fp_t ifce_width,
                       const fp_t dx, const fp_t dy, const fp_t dz,
                       const fp_t dt)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);
	nucleation_kernel<<<num_tiles, tile_size, 0, stream>>> (
	    dev->conc_Cr_new, dev->conc_Nb_new,
	    dev->phi_del_new, dev->phi_lav_new,
        dev->prng,
	    nx, ny, nm,
	    D_Cr[0], D_Nb[1],
	    sigma_del, sigma_lav,
	    lattice_const, ifce_width,
	    dx, dy, dz, dt);
}

__global__ void nickel_kernel(fp_t* d_conc_Cr, fp_t* d_conc_Nb, fp_t* d_conc_Ni,
                              const int nx, const int ny)
{
	const int thr_x = threadIdx.x;
	const int thr_y = threadIdx.y;
	const int x = blockDim.x * blockIdx.x + thr_x;
	const int y = blockDim.y * blockIdx.y + thr_y;
	const int idx = nx * y + x;

	if (x < nx && y < ny)
		d_conc_Ni[idx] = 1. - d_conc_Cr[idx] - d_conc_Nb[idx]; 
}

void device_compute_Ni(hipStream_t& stream,
                       struct CudaData* dev, struct HostData* host,
                       const int nx, const int ny, const int nm,
                       const int bx, const int by)
{
	/* divide matrices into blocks of bx * by threads */
	dim3 tile_size(bx, by, 1);
	dim3 num_tiles(nTiles(nx, tile_size.x, nm),
	               nTiles(ny, tile_size.y, nm),
	               1);

	nickel_kernel<<<num_tiles, tile_size, 0, stream>>>(
	    dev->conc_Cr_old, dev->conc_Nb_old, dev->conc_Ni,
	    nx, ny);

    hipMemcpyAsync(host->conc_Ni[0], dev->conc_Ni, nx * ny * sizeof(fp_t),
                    hipMemcpyDeviceToHost, stream);
}

void read_out_result(struct CudaData* dev, struct HostData* host, const int nx, const int ny)
{
	hipMemcpy(host->conc_Cr_new[0], dev->conc_Cr_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->conc_Nb_new[0], dev->conc_Nb_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_del_new[0], dev->phi_del_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->phi_lav_new[0], dev->phi_lav_old, nx * ny * sizeof(fp_t),
	           hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Cr[0], dev->gam_Cr, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
	hipMemcpy(host->gam_Nb[0], dev->gam_Nb, nx * ny * sizeof(fp_t), hipMemcpyDeviceToHost);
}
